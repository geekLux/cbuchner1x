#include "hip/hip_runtime.h"
// Auf QuarkCoin spezialisierte Version von Groestl

#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <memory.h>

// it's unfortunate that this is a compile time constant.
#define MAXWELL_OR_FERMI 1

// aus cpu-miner.c
extern int device_map[8];

// aus heavy.cu
extern hipError_t MyStreamSynchronize(hipStream_t stream, int situation, int thr_id);

// Folgende Definitionen sp�ter durch header ersetzen
typedef unsigned char uint8_t;
typedef unsigned int uint32_t;
typedef unsigned long long uint64_t;

// diese Struktur wird in der Init Funktion angefordert
static hipDeviceProp_t props[8];

#define SPH_C32(x)    ((uint32_t)(x ## U))
#define SPH_T32(x)    ((x) & SPH_C32(0xFFFFFFFF))

#define PC32up(j, r)   ((uint32_t)((j) + (r)))
#define PC32dn(j, r)   0
#define QC32up(j, r)   0xFFFFFFFF
#define QC32dn(j, r)   (((uint32_t)(r) << 24) ^ SPH_T32(~((uint32_t)(j) << 24)))

#define B32_0(x)    __byte_perm(x, 0, 0x4440)
//((x) & 0xFF)
#define B32_1(x)    __byte_perm(x, 0, 0x4441)
//(((x) >> 8) & 0xFF)
#define B32_2(x)    __byte_perm(x, 0, 0x4442)
//(((x) >> 16) & 0xFF)
#define B32_3(x)    __byte_perm(x, 0, 0x4443)
//((x) >> 24)

#if MAXWELL_OR_FERMI
#define USE_SHARED 1
// Maxwell and Fermi cards get the best speed with SHARED access it seems.
#if USE_SHARED
#define T0up(x) (*((uint32_t*)mixtabs + (    (x))))
#define T0dn(x) (*((uint32_t*)mixtabs + (256+(x))))
#define T1up(x) (*((uint32_t*)mixtabs + (512+(x))))
#define T1dn(x) (*((uint32_t*)mixtabs + (768+(x))))
#define T2up(x) (*((uint32_t*)mixtabs + (1024+(x))))
#define T2dn(x) (*((uint32_t*)mixtabs + (1280+(x))))
#define T3up(x) (*((uint32_t*)mixtabs + (1536+(x))))
#define T3dn(x) (*((uint32_t*)mixtabs + (1792+(x))))
#else
#define T0up(x) tex1Dfetch(t0up1, x)
#define T0dn(x) tex1Dfetch(t0dn1, x)
#define T1up(x) tex1Dfetch(t1up1, x)
#define T1dn(x) tex1Dfetch(t1dn1, x)
#define T2up(x) tex1Dfetch(t2up1, x)
#define T2dn(x) tex1Dfetch(t2dn1, x)
#define T3up(x) tex1Dfetch(t3up1, x)
#define T3dn(x) tex1Dfetch(t3dn1, x)
#endif
#else
#define USE_SHARED 1
// a healthy mix between shared and textured access provides the highest speed on Compute 3.0 and 3.5!
#define T0up(x) (*((uint32_t*)mixtabs + (    (x))))
#define T0dn(x) tex1Dfetch(t0dn1, x)
#define T1up(x) tex1Dfetch(t1up1, x)
#define T1dn(x) (*((uint32_t*)mixtabs + (768+(x))))
#define T2up(x) tex1Dfetch(t2up1, x)
#define T2dn(x) (*((uint32_t*)mixtabs + (1280+(x))))
#define T3up(x) (*((uint32_t*)mixtabs + (1536+(x))))
#define T3dn(x) tex1Dfetch(t3dn1, x)
#endif

texture<unsigned int, 1, hipReadModeElementType> t0up1;
texture<unsigned int, 1, hipReadModeElementType> t0dn1;
texture<unsigned int, 1, hipReadModeElementType> t1up1;
texture<unsigned int, 1, hipReadModeElementType> t1dn1;
texture<unsigned int, 1, hipReadModeElementType> t2up1;
texture<unsigned int, 1, hipReadModeElementType> t2dn1;
texture<unsigned int, 1, hipReadModeElementType> t3up1;
texture<unsigned int, 1, hipReadModeElementType> t3dn1;

extern uint32_t T0up_cpu[];
extern uint32_t T0dn_cpu[];
extern uint32_t T1up_cpu[];
extern uint32_t T1dn_cpu[];
extern uint32_t T2up_cpu[];
extern uint32_t T2dn_cpu[];
extern uint32_t T3up_cpu[];
extern uint32_t T3dn_cpu[];

__device__ __forceinline__ void quark_groestl512_perm_P(uint32_t *a, char *mixtabs)
{
    uint32_t t[32];

//#pragma unroll 14
    for(int r=0;r<14;r++)
    {
        switch(r)
        {
            case 0:
#pragma unroll 16
                for(int k=0;k<16;k++) a[(k*2)+0] ^= PC32up(k<< 4, 0); break;
            case 1:
#pragma unroll 16
                for(int k=0;k<16;k++) a[(k*2)+0] ^= PC32up(k<< 4, 1); break;
            case 2:
#pragma unroll 16
                for(int k=0;k<16;k++) a[(k*2)+0] ^= PC32up(k<< 4, 2); break;
            case 3:
#pragma unroll 16
                for(int k=0;k<16;k++) a[(k*2)+0] ^= PC32up(k<< 4, 3); break;
            case 4:
#pragma unroll 16
                for(int k=0;k<16;k++) a[(k*2)+0] ^= PC32up(k<< 4, 4); break;
            case 5:
#pragma unroll 16
                for(int k=0;k<16;k++) a[(k*2)+0] ^= PC32up(k<< 4, 5); break;
            case 6:
#pragma unroll 16
                for(int k=0;k<16;k++) a[(k*2)+0] ^= PC32up(k<< 4, 6); break;
            case 7:
#pragma unroll 16
                for(int k=0;k<16;k++) a[(k*2)+0] ^= PC32up(k<< 4, 7); break;
            case 8:
#pragma unroll 16
                for(int k=0;k<16;k++) a[(k*2)+0] ^= PC32up(k<< 4, 8); break;
            case 9:
#pragma unroll 16
                for(int k=0;k<16;k++) a[(k*2)+0] ^= PC32up(k<< 4, 9); break;
            case 10:
#pragma unroll 16
                for(int k=0;k<16;k++) a[(k*2)+0] ^= PC32up(k<< 4, 10); break;
            case 11:
#pragma unroll 16
                for(int k=0;k<16;k++) a[(k*2)+0] ^= PC32up(k<< 4, 11); break;
            case 12:
#pragma unroll 16
                for(int k=0;k<16;k++) a[(k*2)+0] ^= PC32up(k<< 4, 12); break;
            case 13:
#pragma unroll 16
                for(int k=0;k<16;k++) a[(k*2)+0] ^= PC32up(k<< 4, 13); break;
        }

        // RBTT
#pragma unroll 16
        for(int k=0;k<32;k+=2)
        {
            uint32_t t0_0 = B32_0(a[(k     ) & 0x1f]), t9_0  = B32_0(a[(k +  9) & 0x1f]);
            uint32_t t2_1 = B32_1(a[(k +  2) & 0x1f]), t11_1 = B32_1(a[(k + 11) & 0x1f]);
            uint32_t t4_2 = B32_2(a[(k +  4) & 0x1f]), t13_2 = B32_2(a[(k + 13) & 0x1f]);
            uint32_t t6_3 = B32_3(a[(k +  6) & 0x1f]), t23_3 = B32_3(a[(k + 23) & 0x1f]);
	    
            t[k + 0] =  T0up( t0_0 ) ^ T1up(  t2_1 ) ^ T2up(  t4_2 ) ^ T3up(  t6_3 ) ^ 
                        T0dn( t9_0 ) ^ T1dn( t11_1 ) ^ T2dn( t13_2 ) ^ T3dn( t23_3 );

            t[k + 1] =  T0dn( t0_0 ) ^ T1dn(  t2_1 ) ^ T2dn(  t4_2 ) ^ T3dn(  t6_3 ) ^ 
                        T0up( t9_0 ) ^ T1up( t11_1 ) ^ T2up( t13_2 ) ^ T3up( t23_3 );
        }
#pragma unroll 32
        for(int k=0;k<32;k++)
            a[k] = t[k];
    }
}

__device__ __forceinline__ void quark_groestl512_perm_Q(uint32_t *a, char *mixtabs)
{    
//#pragma unroll 14
    for(int r=0;r<14;r++)
    {
        uint32_t t[32];

        switch(r)
        {
            case 0:
    #pragma unroll 16
                for(int k=0;k<16;k++) { a[(k*2)+0] ^= QC32up(k<< 4, 0); a[(k*2)+1] ^= QC32dn(k<< 4, 0);} break;
            case 1:
    #pragma unroll 16
                for(int k=0;k<16;k++) { a[(k*2)+0] ^= QC32up(k<< 4, 1); a[(k*2)+1] ^= QC32dn(k<< 4, 1);} break;
            case 2:
    #pragma unroll 16
                for(int k=0;k<16;k++) { a[(k*2)+0] ^= QC32up(k<< 4, 2); a[(k*2)+1] ^= QC32dn(k<< 4, 2);} break;
            case 3:
    #pragma unroll 16
                for(int k=0;k<16;k++) { a[(k*2)+0] ^= QC32up(k<< 4, 3); a[(k*2)+1] ^= QC32dn(k<< 4, 3);} break;
            case 4:
    #pragma unroll 16
                for(int k=0;k<16;k++) { a[(k*2)+0] ^= QC32up(k<< 4, 4); a[(k*2)+1] ^= QC32dn(k<< 4, 4);} break;
            case 5:
    #pragma unroll 16
                for(int k=0;k<16;k++) { a[(k*2)+0] ^= QC32up(k<< 4, 5); a[(k*2)+1] ^= QC32dn(k<< 4, 5);} break;
            case 6:
    #pragma unroll 16
                for(int k=0;k<16;k++) { a[(k*2)+0] ^= QC32up(k<< 4, 6); a[(k*2)+1] ^= QC32dn(k<< 4, 6);} break;
            case 7:
    #pragma unroll 16
                for(int k=0;k<16;k++) { a[(k*2)+0] ^= QC32up(k<< 4, 7); a[(k*2)+1] ^= QC32dn(k<< 4, 7);} break;
            case 8:
    #pragma unroll 16
                for(int k=0;k<16;k++) { a[(k*2)+0] ^= QC32up(k<< 4, 8); a[(k*2)+1] ^= QC32dn(k<< 4, 8);} break;
            case 9:
    #pragma unroll 16
                for(int k=0;k<16;k++) { a[(k*2)+0] ^= QC32up(k<< 4, 9); a[(k*2)+1] ^= QC32dn(k<< 4, 9);} break;
            case 10:
    #pragma unroll 16
                for(int k=0;k<16;k++) { a[(k*2)+0] ^= QC32up(k<< 4, 10); a[(k*2)+1] ^= QC32dn(k<< 4, 10);} break;
            case 11:
    #pragma unroll 16
                for(int k=0;k<16;k++) { a[(k*2)+0] ^= QC32up(k<< 4, 11); a[(k*2)+1] ^= QC32dn(k<< 4, 11);} break;
            case 12:
    #pragma unroll 16
                for(int k=0;k<16;k++) { a[(k*2)+0] ^= QC32up(k<< 4, 12); a[(k*2)+1] ^= QC32dn(k<< 4, 12);} break;
            case 13:
    #pragma unroll 16
                for(int k=0;k<16;k++) { a[(k*2)+0] ^= QC32up(k<< 4, 13); a[(k*2)+1] ^= QC32dn(k<< 4, 13);} break;
        }

        // RBTT
#pragma unroll 16
        for(int k=0;k<32;k+=2)
        {
            uint32_t t2_0  = B32_0(a[(k +  2) & 0x1f]), t1_0  = B32_0(a[(k +  1) & 0x1f]);
            uint32_t t6_1  = B32_1(a[(k +  6) & 0x1f]), t5_1  = B32_1(a[(k +  5) & 0x1f]);
            uint32_t t10_2 = B32_2(a[(k + 10) & 0x1f]), t9_2  = B32_2(a[(k +  9) & 0x1f]);
            uint32_t t22_3 = B32_3(a[(k + 22) & 0x1f]), t13_3 = B32_3(a[(k + 13) & 0x1f]);
	    
            t[k + 0] =  T0up( t2_0 ) ^ T1up( t6_1 ) ^ T2up( t10_2 ) ^ T3up( t22_3 ) ^ 
                        T0dn( t1_0 ) ^ T1dn( t5_1 ) ^ T2dn(  t9_2 ) ^ T3dn( t13_3 );

            t[k + 1] =  T0dn( t2_0 ) ^ T1dn( t6_1 ) ^ T2dn( t10_2 ) ^ T3dn( t22_3 ) ^ 
                        T0up( t1_0 ) ^ T1up( t5_1 ) ^ T2up(  t9_2 ) ^ T3up( t13_3 );
        }
#pragma unroll 32
        for(int k=0;k<32;k++)
            a[k] = t[k];
    }
}
__global__ void  quark_groestl512_gpu_hash_64(int threads, uint32_t startNounce, uint32_t *g_hash, uint32_t *g_nonceVector)
{
#if USE_SHARED
    extern __shared__ char mixtabs[];

    if (threadIdx.x < 256)
    {
        *((uint32_t*)mixtabs + (    threadIdx.x)) = tex1Dfetch(t0up1, threadIdx.x);
        *((uint32_t*)mixtabs + (256+threadIdx.x)) = tex1Dfetch(t0dn1, threadIdx.x);
        *((uint32_t*)mixtabs + (512+threadIdx.x)) = tex1Dfetch(t1up1, threadIdx.x);
        *((uint32_t*)mixtabs + (768+threadIdx.x)) = tex1Dfetch(t1dn1, threadIdx.x);
        *((uint32_t*)mixtabs + (1024+threadIdx.x)) = tex1Dfetch(t2up1, threadIdx.x);
        *((uint32_t*)mixtabs + (1280+threadIdx.x)) = tex1Dfetch(t2dn1, threadIdx.x);
        *((uint32_t*)mixtabs + (1536+threadIdx.x)) = tex1Dfetch(t3up1, threadIdx.x);
        *((uint32_t*)mixtabs + (1792+threadIdx.x)) = tex1Dfetch(t3dn1, threadIdx.x);
    }

    __syncthreads();
#endif

    int thread = (blockDim.x * blockIdx.x + threadIdx.x);
    if (thread < threads)
    {
        // GROESTL
        uint32_t message[32];
        uint32_t state[32];

        uint32_t nounce = (g_nonceVector != NULL) ? g_nonceVector[thread] : (startNounce + thread);

        int hashPosition = nounce - startNounce;
        uint32_t *inpHash = &g_hash[16 * hashPosition];

#pragma unroll 16
        for(int k=0;k<16;k++) message[k] = inpHash[k];
#pragma unroll 14
        for(int k=1;k<15;k++)
            message[k+16] = 0;

        message[16] = 0x80;
        message[31] = 0x01000000;

#pragma unroll 32
        for(int u=0;u<32;u++) state[u] = message[u];
        state[31] ^= 0x20000;

        // Perm
#if USE_SHARED
        quark_groestl512_perm_P(state, mixtabs);
        state[31] ^= 0x20000;
        quark_groestl512_perm_Q(message, mixtabs);
#else
        quark_groestl512_perm_P(state, NULL);
        state[31] ^= 0x20000;
        quark_groestl512_perm_Q(message, NULL);
#endif
#pragma unroll 32
        for(int u=0;u<32;u++) state[u] ^= message[u];

#pragma unroll 32
        for(int u=0;u<32;u++) message[u] = state[u];

#if USE_SHARED
        quark_groestl512_perm_P(message, mixtabs);
#else
        quark_groestl512_perm_P(message, NULL);
#endif

#pragma unroll 32
        for(int u=0;u<32;u++) state[u] ^= message[u];
        // Erzeugten Hash rausschreiben

        uint32_t *outpHash = &g_hash[16 * hashPosition];

#pragma unroll 16
        for(int k=0;k<16;k++) outpHash[k] = state[k+16];
    }
}

#define texDef(texname, texmem, texsource, texsize) \
    unsigned int *texmem; \
    hipMalloc(&texmem, texsize); \
    hipMemcpy(texmem, texsource, texsize, hipMemcpyHostToDevice); \
    texname.normalized = 0; \
    texname.filterMode = hipFilterModePoint; \
    texname.addressMode[0] = hipAddressModeClamp; \
    { hipChannelFormatDesc channelDesc = hipCreateChannelDesc<unsigned int>(); \
      hipBindTexture(NULL, &texname, texmem, &channelDesc, texsize ); } \

// Setup-Funktionen
__host__ void quark_groestl512_cpu_init(int thr_id, int threads)
{
    hipGetDeviceProperties(&props[thr_id], device_map[thr_id]);

// Texturen mit obigem Makro initialisieren
    texDef(t0up1, d_T0up, T0up_cpu, sizeof(uint32_t)*256);
    texDef(t0dn1, d_T0dn, T0dn_cpu, sizeof(uint32_t)*256);
    texDef(t1up1, d_T1up, T1up_cpu, sizeof(uint32_t)*256);
    texDef(t1dn1, d_T1dn, T1dn_cpu, sizeof(uint32_t)*256);
    texDef(t2up1, d_T2up, T2up_cpu, sizeof(uint32_t)*256);
    texDef(t2dn1, d_T2dn, T2dn_cpu, sizeof(uint32_t)*256);
    texDef(t3up1, d_T3up, T3up_cpu, sizeof(uint32_t)*256);
    texDef(t3dn1, d_T3dn, T3dn_cpu, sizeof(uint32_t)*256);
}

__host__ void quark_groestl512_cpu_hash_64(int thr_id, int threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash, int order)
{
    // Compute 3.5 und 5.x Ger�te am besten mit 768 Threads ansteuern,
    // alle anderen mit 512 Threads.
    int threadsperblock = ((props[thr_id].major == 3 && props[thr_id].minor == 5) || props[thr_id].major > 3) ? 768 : 512;

    // berechne wie viele Thread Blocks wir brauchen
    dim3 grid((threads + threadsperblock-1)/threadsperblock);
    dim3 block(threadsperblock);

    // Gr��e des dynamischen Shared Memory Bereichs
#if USE_SHARED
    size_t shared_size = 8 * 256 * sizeof(uint32_t);
#else
    size_t shared_size = 0;
#endif

//    fprintf(stderr, "threads=%d, %d blocks, %d threads per block, %d bytes shared\n", threads, grid.x, block.x, shared_size);
    //fprintf(stderr, "ThrID: %d\n", thr_id);
    quark_groestl512_gpu_hash_64<<<grid, block, shared_size>>>(threads, startNounce, d_hash, d_nonceVector);

    // Strategisches Sleep Kommando zur Senkung der CPU Last
    MyStreamSynchronize(NULL, order, thr_id);
}

__host__ void quark_doublegroestl512_cpu_hash_64(int thr_id, int threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash, int order)
{
    // Compute 3.5 und 5.x Ger�te am besten mit 768 Threads ansteuern,
    // alle anderen mit 512 Threads.
    int threadsperblock = ((props[thr_id].major == 3 && props[thr_id].minor == 5) || props[thr_id].major > 3) ? 768 : 512;

    // berechne wie viele Thread Blocks wir brauchen
    dim3 grid((threads + threadsperblock-1)/threadsperblock);
    dim3 block(threadsperblock);

    // Gr��e des dynamischen Shared Memory Bereichs
#if USE_SHARED
    size_t shared_size = 8 * 256 * sizeof(uint32_t);
#else
    size_t shared_size = 0;
#endif

//  fprintf(stderr, "threads=%d, %d blocks, %d threads per block, %d bytes shared\n", threads, grid.x, block.x, shared_size);
    //fprintf(stderr, "ThrID: %d\n", thr_id);
    quark_groestl512_gpu_hash_64<<<grid, block, shared_size>>>(threads, startNounce, d_hash, d_nonceVector);
    quark_groestl512_gpu_hash_64<<<grid, block, shared_size>>>(threads, startNounce, d_hash, d_nonceVector);

    // Strategisches Sleep Kommando zur Senkung der CPU Last
    MyStreamSynchronize(NULL, order, thr_id);
}
