/* Diese Funktion ist auf 84-Byte gro�e Eingabedaten ausgerichtet (Heavycoin) */
#include <hip/hip_runtime.h>



#include <stdio.h>
#include <memory.h>

#define USE_SHARED 0
#define W_ALIGNMENT 65

// Folgende Definitionen sp�ter durch header ersetzen
typedef unsigned int uint32_t;
typedef unsigned char uint8_t;
typedef unsigned short uint16_t;

// globaler Speicher f�r alle HeftyHashes aller Threads
uint32_t *d_heftyHashes[8];

/* Hash-Tabellen */
__constant__ uint32_t hefty_gpu_constantTable[64];

// muss expandiert werden
__constant__ uint32_t hefty_gpu_blockHeader[16]; // 2x512 Bit Message
__constant__ uint32_t hefty_gpu_register[8];
__constant__ uint32_t hefty_gpu_sponge[4];

uint32_t hefty_cpu_hashTable[] = { 0x6a09e667UL,
    0xbb67ae85UL,
    0x3c6ef372UL,
    0xa54ff53aUL,
    0x510e527fUL,
    0x9b05688cUL,
    0x1f83d9abUL,
    0x5be0cd19UL };
uint32_t hefty_cpu_constantTable[] = {
	0x428a2f98UL, 0x71374491UL, 0xb5c0fbcfUL, 0xe9b5dba5UL,
    0x3956c25bUL, 0x59f111f1UL, 0x923f82a4UL, 0xab1c5ed5UL,
    0xd807aa98UL, 0x12835b01UL, 0x243185beUL, 0x550c7dc3UL,
    0x72be5d74UL, 0x80deb1feUL, 0x9bdc06a7UL, 0xc19bf174UL,
    0xe49b69c1UL, 0xefbe4786UL, 0x0fc19dc6UL, 0x240ca1ccUL,
    0x2de92c6fUL, 0x4a7484aaUL, 0x5cb0a9dcUL, 0x76f988daUL,
    0x983e5152UL, 0xa831c66dUL, 0xb00327c8UL, 0xbf597fc7UL,
    0xc6e00bf3UL, 0xd5a79147UL, 0x06ca6351UL, 0x14292967UL,
    0x27b70a85UL, 0x2e1b2138UL, 0x4d2c6dfcUL, 0x53380d13UL,
    0x650a7354UL, 0x766a0abbUL, 0x81c2c92eUL, 0x92722c85UL,
    0xa2bfe8a1UL, 0xa81a664bUL, 0xc24b8b70UL, 0xc76c51a3UL,
    0xd192e819UL, 0xd6990624UL, 0xf40e3585UL, 0x106aa070UL,
    0x19a4c116UL, 0x1e376c08UL, 0x2748774cUL, 0x34b0bcb5UL,
    0x391c0cb3UL, 0x4ed8aa4aUL, 0x5b9cca4fUL, 0x682e6ff3UL,
    0x748f82eeUL, 0x78a5636fUL, 0x84c87814UL, 0x8cc70208UL,
    0x90befffaUL, 0xa4506cebUL, 0xbef9a3f7UL, 0xc67178f2UL
};

#define S(x, n)			(((x) >> (n)) | ((x) << (32 - (n))))
#define R(x, n)			((x) >> (n))
#define Ch(x, y, z)		((x & (y ^ z)) ^ z)
#define Maj(x, y, z)	((x & (y | z)) | (y & z))
#define S0(x)			(S(x, 2) ^ S(x, 13) ^ S(x, 22))
#define S1(x)			(S(x, 6) ^ S(x, 11) ^ S(x, 25))
#define s0(x)			(S(x, 7) ^ S(x, 18) ^ R(x, 3))
#define s1(x)			(S(x, 17) ^ S(x, 19) ^ R(x, 10))

#define SWAB32(x)		( ((x & 0x000000FF) << 24) | ((x & 0x0000FF00) << 8) | ((x & 0x00FF0000) >> 8) | ((x & 0xFF000000) >> 24) )

// uint8_t
#define smoosh4(x)		( ((x)>>4) ^ ((x) & 0x0F) )
__host__ __forceinline__ __device__ uint8_t smoosh2(uint32_t x)
{
	uint16_t w = (x >> 16) ^ (x & 0xffff);
	uint8_t n = smoosh4( (uint8_t)( (w >> 8) ^ (w & 0xFF) ) );
	return (n >> 2) ^ (n & 0x03);
}
// 4 auf einmal
#define smoosh4Quad(x)	( (((x)>>4) ^ (x)) & 0x0F0F0F0F )
#define getByte(x,y)	( ((x) >> (y)) & 0xFF )

__host__ __device__ void Mangle(uint32_t *inp)
{
	uint32_t r = smoosh4Quad(inp[0]);
	//uint8_t r0 = smoosh4( (uint8_t)(inp[0] >> 24) );
	//uint8_t r1 = smoosh4( (uint8_t)(inp[0] >> 16) );
	//uint8_t r2 = smoosh4( (uint8_t)(inp[0] >> 8) );
	//uint8_t r3 = smoosh4( (uint8_t)(inp[0] & 0xFF) );

	inp[1] = inp[1] ^ S(inp[0], getByte(r, 24));
	
	switch (smoosh2(inp[1])) {
      case 0: inp[2] ^= S(inp[0], 1 + getByte(r,24)); break;
      case 1: inp[2] += S(~inp[0], 1 + getByte(r,16)); break;
      case 2: inp[2] &= S(~inp[0], 1 + getByte(r,8)); break;
      case 3: inp[2] ^= S(inp[0], 1 + getByte(r,0)); break;
    }
    
	uint32_t tmp = smoosh2(inp[1] ^ inp[2]);
    switch (tmp) {
      case 0: inp[3] ^= S(inp[0], 2 + getByte(r,24)); break;
      case 1: inp[3] += S(~inp[0], 2 + getByte(r,16)); break;
      case 2: inp[3] &= S(~inp[0], 2 + getByte(r,8)); break;
      case 3: inp[3] ^= S(inp[0], 2 + getByte(r,0)); break;
    }

	inp[0] ^= (inp[1] ^ inp[2]) + inp[3];
}

__host__ __forceinline__ __device__ void Absorb(uint32_t *inp, uint32_t x)
{
	inp[0] ^= x;
	Mangle(inp);
}

__host__ __forceinline__ __device__ uint32_t Squeeze(uint32_t *inp)
{
	uint32_t y = inp[0];
	Mangle(inp);
	return y;
}

__host__ __forceinline__ __device__ uint32_t Br(uint32_t *sponge, uint32_t x)
{
	uint32_t r = Squeeze(sponge);

	//uint8_t r0 = r >> 8;
	uint8_t r1 = r & 0xFF;
	uint32_t y = 1 << ((r >> 8) & 0x1F);

	//uint32_t retVal;
	//retVal = x;
	
	uint32_t resArr[4];
	resArr[0] = x;
	resArr[1] = x & ~y;
	resArr[2] = x | y;
	resArr[3] = x ^ y;
	return resArr[r1 & 0x03];

	/*
	switch(r1 & 0x03)
	{
	case 0:
		break;
    case 1:
        retVal = x & ~y;
		break;
    case 2:
        retVal = x | y;
		break;
    case 3:
        retVal = x ^ y;
		break;
    }
	return retVal;
	*/
}

__forceinline__ __device__ void hefty_gpu_round(uint32_t *regs, uint32_t W, uint32_t K, uint32_t *sponge)
{
	uint32_t tmpBr;

	uint32_t brG = Br(sponge, regs[6]);	
	uint32_t brF = Br(sponge, regs[5]);
	uint32_t tmp1 = Ch(regs[4], brF, brG) + regs[7] + W + K;
	uint32_t brE = Br(sponge, regs[4]);
	uint32_t tmp2 = tmp1 + S1(brE);
	uint32_t brC = Br(sponge, regs[2]);
	uint32_t brB = Br(sponge, regs[1]);
	uint32_t brA = Br(sponge, regs[0]);
	uint32_t tmp3 = Maj(brA, brB, brC);
	tmpBr = Br(sponge, regs[0]);
	uint32_t tmp4 = tmp3 + S0(tmpBr);
	tmpBr = Br(sponge, tmp2);

	#pragma unroll 7
	for (int k=6; k >= 0; k--) regs[k+1] = regs[k];
	regs[0] = tmp2 + tmp4;
	regs[4] += tmpBr;
}

__host__ void hefty_cpu_round(uint32_t *regs, uint32_t W, uint32_t K, uint32_t *sponge)
{
	uint32_t tmpBr;

	uint32_t brG = Br(sponge, regs[6]);	
	uint32_t brF = Br(sponge, regs[5]);
	uint32_t tmp1 = Ch(regs[4], brF, brG) + regs[7] + W + K;
	uint32_t brE = Br(sponge, regs[4]);
	uint32_t tmp2 = tmp1 + S1(brE);
	uint32_t brC = Br(sponge, regs[2]);
	uint32_t brB = Br(sponge, regs[1]);
	uint32_t brA = Br(sponge, regs[0]);
	uint32_t tmp3 = Maj(brA, brB, brC);
	tmpBr = Br(sponge, regs[0]);
	uint32_t tmp4 = tmp3 + S0(tmpBr);
	tmpBr = Br(sponge, tmp2);

	for (int k=6; k >= 0; k--) regs[k+1] = regs[k];
	regs[0] = tmp2 + tmp4;
	regs[4] += tmpBr;
}

// Die Hash-Funktion
__global__ void hefty_gpu_hash(int threads, uint32_t startNounce, void *outputHash)
{
	int thread = (blockDim.x * blockIdx.x + threadIdx.x);
	if (thread < threads)
	{
		// bestimme den aktuellen Z�hler
		uint32_t nounce = startNounce + thread;
	
		// jeder thread in diesem  Block bekommt sein eigenes W Array im Shared memory
#if USE_SHARED
		extern __shared__ unsigned char s[];
		uint32_t *W = (uint32_t *)(&s[W_ALIGNMENT * sizeof(uint32_t) * threadIdx.x]);
#else
		// reduktion von 256 byte auf 128 byte
		uint32_t W1[16];
		uint32_t W2[16];
#endif

		// Initialisiere die register a bis h mit der Hash-Tabelle
		uint32_t regs[8];
		uint32_t hash[8];
		uint32_t sponge[4];
	
#pragma unroll 4
		for(int k=0; k < 4; k++)
			sponge[k] = hefty_gpu_sponge[k];

		// pre
#pragma unroll 8
		for (int k=0; k < 8; k++)
		{
			regs[k] = hefty_gpu_register[k];
			hash[k] = regs[k];
		}
	
		//memcpy(W, &hefty_gpu_blockHeader[0], sizeof(uint32_t) * 16); // verbleibende 20 bytes aus Block 2 plus padding
#pragma unroll 16
		for(int k=0;k<16;k++)
			W1[k] = hefty_gpu_blockHeader[k];
		W1[3] = SWAB32(nounce);


		// 2. Runde
#pragma unroll 16
		for(int j=0;j<16;j++)
			Absorb(sponge, W1[j] ^ hefty_gpu_constantTable[j]);

// Progress W1 (Bytes 0...63)
#pragma unroll 16
		for(int j=0;j<16;j++)
		{
			Absorb(sponge, regs[3] ^ regs[7]);
			hefty_gpu_round(regs, W1[j], hefty_gpu_constantTable[j], sponge);
		}

// Progress W2 (Bytes 64...127) then W3 (Bytes 128...191) ...
		
#pragma unroll 3
		for(int k=0;k<3;k++)
		{
	#pragma unroll 2
			for(int j=0;j<2;j++)
				W2[j] = s1(W1[14+j]) + W1[9+j] + s0(W1[1+j]) + W1[j];
	#pragma unroll 5
			for(int j=2;j<7;j++)
				W2[j] = s1(W2[j-2]) + W1[9+j] + s0(W1[1+j]) + W1[j];

	#pragma unroll 8
			for(int j=7;j<15;j++)
				W2[j] = s1(W2[j-2]) + W2[j-7] + s0(W1[1+j]) + W1[j];

			W2[15] = s1(W2[13]) + W2[8] + s0(W2[0]) + W1[15];

	#pragma unroll 16
			for(int j=0;j<16;j++)
			{
				Absorb(sponge, regs[3] + regs[7]);
				hefty_gpu_round(regs, W2[j], hefty_gpu_constantTable[j + 16 * (k+1)], sponge);
			}
	#pragma unroll 16
			for(int j=0;j<16;j++)
				W1[j] = W2[j];
		}
		

#pragma unroll 8
		for(int k=0;k<8;k++)
			hash[k] += regs[k];

#pragma unroll 8
		for(int k=0;k<8;k++)
			((uint32_t*)outputHash)[8*thread+k] = SWAB32(hash[k]);
	}
}

// Setup-Funktionen
__host__ void hefty_cpu_init(int thr_id, int threads)
{
	hipSetDevice(thr_id);

	// Kopiere die Hash-Tabellen in den GPU-Speicher
	hipMemcpyToSymbol(	HIP_SYMBOL(hefty_gpu_constantTable),
						hefty_cpu_constantTable,
						sizeof(uint32_t) * 64 );

	// Speicher f�r alle Hefty1 hashes belegen
	hipMalloc(&d_heftyHashes[thr_id], 8 * sizeof(uint32_t) * threads);
}

__host__ void hefty_cpu_setBlock(int thr_id, int threads, void *data)
	// data muss 84-Byte haben!
{
	// Nachricht expandieren und setzen
	uint32_t msgBlock[32];

	memset(msgBlock, 0, sizeof(uint32_t) * 32);
	memcpy(&msgBlock[0], data, 84);
	msgBlock[21] |= 0x80;
	msgBlock[31] = 672; // bitlen
	
	for(int i=0;i<31;i++) // Byteorder drehen
		msgBlock[i] = SWAB32(msgBlock[i]);

	// die erste Runde wird auf der CPU durchgef�hrt, da diese f�r
	// alle Threads gleich ist. Der Hash wird dann an die Threads
	// �bergeben

	// Erstelle expandierten Block W
	uint32_t W[64];	
	memcpy(W, &msgBlock[0], sizeof(uint32_t) * 16);	
	for(int j=16;j<64;j++)
		W[j] = s1(W[j-2]) + W[j-7] + s0(W[j-15]) + W[j-16];

	// Initialisiere die register a bis h mit der Hash-Tabelle
	uint32_t regs[8];
	uint32_t hash[8];
	uint32_t sponge[4];

	// pre
	memset(sponge, 0, sizeof(uint32_t) * 4);
    for (int k=0; k < 8; k++)
	{
		regs[k] = hefty_cpu_hashTable[k];
		hash[k] = regs[k];
	}	

	// 1. Runde
	for(int j=0;j<16;j++)
		Absorb(sponge, W[j] ^ hefty_cpu_constantTable[j]);

	for(int j=0;j<16;j++)
	{
		Absorb(sponge, regs[3] ^ regs[7]);
		hefty_cpu_round(regs, W[j], hefty_cpu_constantTable[j], sponge);
	}

	for(int j=16;j<64;j++)
	{
		Absorb(sponge, regs[3] + regs[7]);
		hefty_cpu_round(regs, W[j], hefty_cpu_constantTable[j], sponge);
	}

	for(int k=0;k<8;k++)
		hash[k] += regs[k];

	// sponge speichern

	hipMemcpyToSymbol(	HIP_SYMBOL(hefty_gpu_sponge),
						sponge,
						sizeof(uint32_t) * 4 );
	// hash speichern
	hipMemcpyToSymbol(	HIP_SYMBOL(hefty_gpu_register),
						hash,
						sizeof(uint32_t) * 8 );

	// Blockheader setzen (korrekte Nonce fehlt da drin noch)
	hipMemcpyToSymbol(	HIP_SYMBOL(hefty_gpu_blockHeader),
						&msgBlock[16],
						64);
}

__host__ void hefty_cpu_hash(int thr_id, int threads, int startNounce)
{
	const int threadsperblock = 128;

	// berechne wie viele Thread Blocks wir brauchen
	dim3 grid((threads + threadsperblock-1)/threadsperblock);
	dim3 block(threadsperblock);

	// Gr��e des dynamischen Shared Memory Bereichs (abh�ngig von der Threadanzahl)
#if USE_SHARED
	size_t shared_size = W_ALIGNMENT*sizeof(uint32_t)*threadsperblock;  // ein uint32_t eingef�gt gegen Bank Konflikte
#else
	size_t shared_size = 0;
#endif

//	fprintf(stderr, "threads=%d, %d blocks, %d threads per block, %d bytes shared\n", threads, grid.x, block.x, shared_size);

	hefty_gpu_hash<<<grid, block, shared_size>>>(threads, startNounce, (void*)d_heftyHashes[thr_id]);
}
