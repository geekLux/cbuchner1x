#include "hip/hip_runtime.h"
// Auf Myriadcoin spezialisierte Version von Groestl

#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <memory.h>

// it's unfortunate that this is a compile time constant.
#define MAXWELL_OR_FERMI 1

// aus cpu-miner.c
extern int device_map[8];

// aus heavy.cu
extern hipError_t MyStreamSynchronize(hipStream_t stream, int situation, int thr_id);

// Folgende Definitionen sp�ter durch header ersetzen
typedef unsigned char uint8_t;
typedef unsigned short uint16_t;
typedef unsigned int uint32_t;

// diese Struktur wird in der Init Funktion angefordert
static hipDeviceProp_t props;

__constant__ uint32_t pTarget[8]; // Single GPU
extern uint32_t *d_resultNonce[8];

__constant__ uint32_t myriadgroestl_gpu_msg[32];

// muss expandiert werden
__constant__ uint32_t myr_sha256_gpu_constantTable[64];
__constant__ uint32_t myr_sha256_gpu_hashTable[8];

uint32_t myr_sha256_cpu_hashTable[] = { 
	0x6a09e667, 0xbb67ae85, 0x3c6ef372, 0xa54ff53a, 0x510e527f, 0x9b05688c, 0x1f83d9ab, 0x5be0cd19 };
uint32_t myr_sha256_cpu_constantTable[] = {
	0x428a2f98, 0x71374491, 0xb5c0fbcf, 0xe9b5dba5, 0x3956c25b, 0x59f111f1, 0x923f82a4, 0xab1c5ed5,
	0xd807aa98, 0x12835b01, 0x243185be, 0x550c7dc3, 0x72be5d74, 0x80deb1fe, 0x9bdc06a7, 0xc19bf174,
	0xe49b69c1, 0xefbe4786, 0x0fc19dc6, 0x240ca1cc, 0x2de92c6f, 0x4a7484aa, 0x5cb0a9dc, 0x76f988da,
	0x983e5152, 0xa831c66d, 0xb00327c8, 0xbf597fc7, 0xc6e00bf3, 0xd5a79147, 0x06ca6351, 0x14292967,
	0x27b70a85, 0x2e1b2138, 0x4d2c6dfc, 0x53380d13, 0x650a7354, 0x766a0abb, 0x81c2c92e, 0x92722c85,
	0xa2bfe8a1, 0xa81a664b, 0xc24b8b70, 0xc76c51a3, 0xd192e819, 0xd6990624, 0xf40e3585, 0x106aa070,
	0x19a4c116, 0x1e376c08, 0x2748774c, 0x34b0bcb5, 0x391c0cb3, 0x4ed8aa4a, 0x5b9cca4f, 0x682e6ff3,
	0x748f82ee, 0x78a5636f, 0x84c87814, 0x8cc70208, 0x90befffa, 0xa4506ceb, 0xbef9a3f7, 0xc67178f2,
};

#if __CUDA_ARCH__ < 350 
    // Kepler (Compute 3.0)
    #define ROTR32(x, n) (((x) >> (n)) | ((x) << (32 - (n))))
#else
    // Kepler (Compute 3.5)
    #define ROTR32(x, n) __funnelshift_r( (x), (x), (n) )
#endif
#define R(x, n)			((x) >> (n))
#define Ch(x, y, z)		((x & (y ^ z)) ^ z)
#define Maj(x, y, z)	((x & (y | z)) | (y & z))
#define S0(x)			(ROTR32(x, 2) ^ ROTR32(x, 13) ^ ROTR32(x, 22))
#define S1(x)			(ROTR32(x, 6) ^ ROTR32(x, 11) ^ ROTR32(x, 25))
#define s0(x)			(ROTR32(x, 7) ^ ROTR32(x, 18) ^ R(x, 3))
#define s1(x)			(ROTR32(x, 17) ^ ROTR32(x, 19) ^ R(x, 10))

#define SWAB32(x)		( ((x & 0x000000FF) << 24) | ((x & 0x0000FF00) << 8) | ((x & 0x00FF0000) >> 8) | ((x & 0xFF000000) >> 24) )

__device__ void myriadgroestl_gpu_sha256(uint32_t *message)
{
	uint32_t W1[16];
	uint32_t W2[16];

	// Initialisiere die register a bis h mit der Hash-Tabelle
	uint32_t regs[8];
	uint32_t hash[8];

	// pre
#pragma unroll 8
	for (int k=0; k < 8; k++)
	{
		regs[k] = myr_sha256_gpu_hashTable[k];
		hash[k] = regs[k];
	}
	
#pragma unroll 16
	for(int k=0;k<16;k++)
		W1[k] = SWAB32(message[k]);

// Progress W1
#pragma unroll 16
	for(int j=0;j<16;j++)
	{
		uint32_t T1, T2;
		T1 = regs[7] + S1(regs[4]) + Ch(regs[4], regs[5], regs[6]) + myr_sha256_gpu_constantTable[j] + W1[j];
		T2 = S0(regs[0]) + Maj(regs[0], regs[1], regs[2]);
		
		#pragma unroll 7
		for (int k=6; k >= 0; k--) regs[k+1] = regs[k];
		regs[0] = T1 + T2;
		regs[4] += T1;
	}

// Progress W2...W3
#pragma unroll 3
	for(int k=0;k<3;k++)
	{
#pragma unroll 2
		for(int j=0;j<2;j++)
			W2[j] = s1(W1[14+j]) + W1[9+j] + s0(W1[1+j]) + W1[j];
#pragma unroll 5
		for(int j=2;j<7;j++)
			W2[j] = s1(W2[j-2]) + W1[9+j] + s0(W1[1+j]) + W1[j];

#pragma unroll 8
		for(int j=7;j<15;j++)
			W2[j] = s1(W2[j-2]) + W2[j-7] + s0(W1[1+j]) + W1[j];

		W2[15] = s1(W2[13]) + W2[8] + s0(W2[0]) + W1[15];

		// Rundenfunktion
#pragma unroll 16
		for(int j=0;j<16;j++)
		{
			uint32_t T1, T2;
			T1 = regs[7] + S1(regs[4]) + Ch(regs[4], regs[5], regs[6]) + myr_sha256_gpu_constantTable[j + 16 * (k+1)] + W2[j];
			T2 = S0(regs[0]) + Maj(regs[0], regs[1], regs[2]);
		
			#pragma unroll 7
			for (int l=6; l >= 0; l--) regs[l+1] = regs[l];
			regs[0] = T1 + T2;
			regs[4] += T1;
		}

#pragma unroll 16
		for(int j=0;j<16;j++)
			W1[j] = W2[j];
	}

#pragma unroll 8
	for(int k=0;k<8;k++)
		hash[k] += regs[k];

	/////
	///// Zweite Runde (wegen Msg-Padding)
	/////
#pragma unroll 8
	for(int k=0;k<8;k++)
		regs[k] = hash[k];

	W1[0] = SWAB32(0x80);
#pragma unroll 14
	for(int k=1;k<15;k++)
		W1[k] = 0;
	W1[15] = 512;

// Progress W1
#pragma unroll 16
	for(int j=0;j<16;j++)
	{
		uint32_t T1, T2;
		T1 = regs[7] + S1(regs[4]) + Ch(regs[4], regs[5], regs[6]) + myr_sha256_gpu_constantTable[j] + W1[j];
		T2 = S0(regs[0]) + Maj(regs[0], regs[1], regs[2]);
		
		#pragma unroll 7
		for (int k=6; k >= 0; k--) regs[k+1] = regs[k];
		regs[0] = T1 + T2;
		regs[4] += T1;
	}

// Progress W2...W3
#pragma unroll 3
	for(int k=0;k<3;k++)
	{
#pragma unroll 2
		for(int j=0;j<2;j++)
			W2[j] = s1(W1[14+j]) + W1[9+j] + s0(W1[1+j]) + W1[j];
#pragma unroll 5
		for(int j=2;j<7;j++)
			W2[j] = s1(W2[j-2]) + W1[9+j] + s0(W1[1+j]) + W1[j];

#pragma unroll 8
		for(int j=7;j<15;j++)
			W2[j] = s1(W2[j-2]) + W2[j-7] + s0(W1[1+j]) + W1[j];

		W2[15] = s1(W2[13]) + W2[8] + s0(W2[0]) + W1[15];

		// Rundenfunktion
#pragma unroll 16
		for(int j=0;j<16;j++)
		{
			uint32_t T1, T2;
			T1 = regs[7] + S1(regs[4]) + Ch(regs[4], regs[5], regs[6]) + myr_sha256_gpu_constantTable[j + 16 * (k+1)] + W2[j];
			T2 = S0(regs[0]) + Maj(regs[0], regs[1], regs[2]);
		
			#pragma unroll 7
			for (int l=6; l >= 0; l--) regs[l+1] = regs[l];
			regs[0] = T1 + T2;
			regs[4] += T1;
		}

#pragma unroll 16
		for(int j=0;j<16;j++)
			W1[j] = W2[j];
	}

#pragma unroll 8
	for(int k=0;k<8;k++)
		hash[k] += regs[k];

	//// FERTIG

#pragma unroll 8
	for(int k=0;k<8;k++)
		message[k] = SWAB32(hash[k]);
}

#define SPH_C32(x)    ((uint32_t)(x ## U))
#define SPH_T32(x)    ((x) & SPH_C32(0xFFFFFFFF))

#define PC32up(j, r)   ((uint32_t)((j) + (r)))
#define PC32dn(j, r)   0
#define QC32up(j, r)   0xFFFFFFFF
#define QC32dn(j, r)   (((uint32_t)(r) << 24) ^ SPH_T32(~((uint32_t)(j) << 24)))

#define B32_0(x)    __byte_perm(x, 0, 0x4440)
//((x) & 0xFF)
#define B32_1(x)    __byte_perm(x, 0, 0x4441)
//(((x) >> 8) & 0xFF)
#define B32_2(x)    __byte_perm(x, 0, 0x4442)
//(((x) >> 16) & 0xFF)
#define B32_3(x)    __byte_perm(x, 0, 0x4443)
//((x) >> 24)

#if MAXWELL_OR_FERMI
#define USE_SHARED 1
// Maxwell and Fermi cards get the best speed with SHARED access it seems.
#if USE_SHARED
#define T0up(x) (*((uint32_t*)mixtabs + (    (x))))
#define T0dn(x) (*((uint32_t*)mixtabs + (256+(x))))
#define T1up(x) (*((uint32_t*)mixtabs + (512+(x))))
#define T1dn(x) (*((uint32_t*)mixtabs + (768+(x))))
#define T2up(x) (*((uint32_t*)mixtabs + (1024+(x))))
#define T2dn(x) (*((uint32_t*)mixtabs + (1280+(x))))
#define T3up(x) (*((uint32_t*)mixtabs + (1536+(x))))
#define T3dn(x) (*((uint32_t*)mixtabs + (1792+(x))))
#else
#define T0up(x) tex1Dfetch(t0up1, x)
#define T0dn(x) tex1Dfetch(t0dn1, x)
#define T1up(x) tex1Dfetch(t1up1, x)
#define T1dn(x) tex1Dfetch(t1dn1, x)
#define T2up(x) tex1Dfetch(t2up1, x)
#define T2dn(x) tex1Dfetch(t2dn1, x)
#define T3up(x) tex1Dfetch(t3up1, x)
#define T3dn(x) tex1Dfetch(t3dn1, x)
#endif
#else
#define USE_SHARED 1
// a healthy mix between shared and textured access provides the highest speed on Compute 3.0 and 3.5!
#define T0up(x) (*((uint32_t*)mixtabs + (    (x))))
#define T0dn(x) tex1Dfetch(t0dn1, x)
#define T1up(x) tex1Dfetch(t1up1, x)
#define T1dn(x) (*((uint32_t*)mixtabs + (768+(x))))
#define T2up(x) tex1Dfetch(t2up1, x)
#define T2dn(x) (*((uint32_t*)mixtabs + (1280+(x))))
#define T3up(x) (*((uint32_t*)mixtabs + (1536+(x))))
#define T3dn(x) tex1Dfetch(t3dn1, x)
#endif

texture<unsigned int, 1, hipReadModeElementType> t0up1;
texture<unsigned int, 1, hipReadModeElementType> t0dn1;
texture<unsigned int, 1, hipReadModeElementType> t1up1;
texture<unsigned int, 1, hipReadModeElementType> t1dn1;
texture<unsigned int, 1, hipReadModeElementType> t2up1;
texture<unsigned int, 1, hipReadModeElementType> t2dn1;
texture<unsigned int, 1, hipReadModeElementType> t3up1;
texture<unsigned int, 1, hipReadModeElementType> t3dn1;

extern uint32_t T0up_cpu[];
extern uint32_t T0dn_cpu[];
extern uint32_t T1up_cpu[];
extern uint32_t T1dn_cpu[];
extern uint32_t T2up_cpu[];
extern uint32_t T2dn_cpu[];
extern uint32_t T3up_cpu[];
extern uint32_t T3dn_cpu[];

#define SWAB32(x)		( ((x & 0x000000FF) << 24) | ((x & 0x0000FF00) << 8) | ((x & 0x00FF0000) >> 8) | ((x & 0xFF000000) >> 24) )


__device__ __forceinline__ void myriadgroestl_perm_P(uint32_t *a, char *mixtabs)
{
	uint32_t t[32];

//#pragma unroll 14
	for(int r=0;r<14;r++)
	{
		switch(r)
		{
			case 0:
#pragma unroll 16
				for(int k=0;k<16;k++) a[(k*2)+0] ^= PC32up(k * 0x10, 0); break;
			case 1:
#pragma unroll 16
				for(int k=0;k<16;k++) a[(k*2)+0] ^= PC32up(k * 0x10, 1); break;
			case 2:
#pragma unroll 16
				for(int k=0;k<16;k++) a[(k*2)+0] ^= PC32up(k * 0x10, 2); break;
			case 3:
#pragma unroll 16
				for(int k=0;k<16;k++) a[(k*2)+0] ^= PC32up(k * 0x10, 3); break;
			case 4:
#pragma unroll 16
				for(int k=0;k<16;k++) a[(k*2)+0] ^= PC32up(k * 0x10, 4); break;
			case 5:
#pragma unroll 16
				for(int k=0;k<16;k++) a[(k*2)+0] ^= PC32up(k * 0x10, 5); break;
			case 6:
#pragma unroll 16
				for(int k=0;k<16;k++) a[(k*2)+0] ^= PC32up(k * 0x10, 6); break;
			case 7:
#pragma unroll 16
				for(int k=0;k<16;k++) a[(k*2)+0] ^= PC32up(k * 0x10, 7); break;
			case 8:
#pragma unroll 16
				for(int k=0;k<16;k++) a[(k*2)+0] ^= PC32up(k * 0x10, 8); break;
			case 9:
#pragma unroll 16
				for(int k=0;k<16;k++) a[(k*2)+0] ^= PC32up(k * 0x10, 9); break;
			case 10:
#pragma unroll 16
				for(int k=0;k<16;k++) a[(k*2)+0] ^= PC32up(k * 0x10, 10); break;
			case 11:
#pragma unroll 16
				for(int k=0;k<16;k++) a[(k*2)+0] ^= PC32up(k * 0x10, 11); break;
			case 12:
#pragma unroll 16
				for(int k=0;k<16;k++) a[(k*2)+0] ^= PC32up(k * 0x10, 12); break;
			case 13:
#pragma unroll 16
				for(int k=0;k<16;k++) a[(k*2)+0] ^= PC32up(k * 0x10, 13); break;
		}

        // RBTT
#pragma unroll 16
        for(int k=0;k<32;k+=2)
        {
            uint32_t t0_0 = B32_0(a[(k     ) & 0x1f]), t9_0  = B32_0(a[(k +  9) & 0x1f]);
            uint32_t t2_1 = B32_1(a[(k +  2) & 0x1f]), t11_1 = B32_1(a[(k + 11) & 0x1f]);
            uint32_t t4_2 = B32_2(a[(k +  4) & 0x1f]), t13_2 = B32_2(a[(k + 13) & 0x1f]);
            uint32_t t6_3 = B32_3(a[(k +  6) & 0x1f]), t23_3 = B32_3(a[(k + 23) & 0x1f]);
        
            t[k + 0] =  T0up( t0_0 ) ^ T1up(  t2_1 ) ^ T2up(  t4_2 ) ^ T3up(  t6_3 ) ^ 
                        T0dn( t9_0 ) ^ T1dn( t11_1 ) ^ T2dn( t13_2 ) ^ T3dn( t23_3 );

            t[k + 1] =  T0dn( t0_0 ) ^ T1dn(  t2_1 ) ^ T2dn(  t4_2 ) ^ T3dn(  t6_3 ) ^ 
                        T0up( t9_0 ) ^ T1up( t11_1 ) ^ T2up( t13_2 ) ^ T3up( t23_3 );
        }
#pragma unroll 32
        for(int k=0;k<32;k++)
            a[k] = t[k];
    }
}

__device__ __forceinline__ void myriadgroestl_perm_Q(uint32_t *a, char *mixtabs)
{	
//#pragma unroll 14
	for(int r=0;r<14;r++)
	{
		uint32_t t[32];

		switch(r)
		{
			case 0:
	#pragma unroll 16
				for(int k=0;k<16;k++) { a[(k*2)+0] ^= QC32up(k * 0x10, 0); a[(k*2)+1] ^= QC32dn(k * 0x10, 0);} break;
			case 1:
	#pragma unroll 16
				for(int k=0;k<16;k++) { a[(k*2)+0] ^= QC32up(k * 0x10, 1); a[(k*2)+1] ^= QC32dn(k * 0x10, 1);} break;
			case 2:
	#pragma unroll 16
				for(int k=0;k<16;k++) { a[(k*2)+0] ^= QC32up(k * 0x10, 2); a[(k*2)+1] ^= QC32dn(k * 0x10, 2);} break;
			case 3:
	#pragma unroll 16
				for(int k=0;k<16;k++) { a[(k*2)+0] ^= QC32up(k * 0x10, 3); a[(k*2)+1] ^= QC32dn(k * 0x10, 3);} break;
			case 4:
	#pragma unroll 16
				for(int k=0;k<16;k++) { a[(k*2)+0] ^= QC32up(k * 0x10, 4); a[(k*2)+1] ^= QC32dn(k * 0x10, 4);} break;
			case 5:
	#pragma unroll 16
				for(int k=0;k<16;k++) { a[(k*2)+0] ^= QC32up(k * 0x10, 5); a[(k*2)+1] ^= QC32dn(k * 0x10, 5);} break;
			case 6:
	#pragma unroll 16
				for(int k=0;k<16;k++) { a[(k*2)+0] ^= QC32up(k * 0x10, 6); a[(k*2)+1] ^= QC32dn(k * 0x10, 6);} break;
			case 7:
	#pragma unroll 16
				for(int k=0;k<16;k++) { a[(k*2)+0] ^= QC32up(k * 0x10, 7); a[(k*2)+1] ^= QC32dn(k * 0x10, 7);} break;
			case 8:
	#pragma unroll 16
				for(int k=0;k<16;k++) { a[(k*2)+0] ^= QC32up(k * 0x10, 8); a[(k*2)+1] ^= QC32dn(k * 0x10, 8);} break;
			case 9:
	#pragma unroll 16
				for(int k=0;k<16;k++) { a[(k*2)+0] ^= QC32up(k * 0x10, 9); a[(k*2)+1] ^= QC32dn(k * 0x10, 9);} break;
			case 10:
	#pragma unroll 16
				for(int k=0;k<16;k++) { a[(k*2)+0] ^= QC32up(k * 0x10, 10); a[(k*2)+1] ^= QC32dn(k * 0x10, 10);} break;
			case 11:
	#pragma unroll 16
				for(int k=0;k<16;k++) { a[(k*2)+0] ^= QC32up(k * 0x10, 11); a[(k*2)+1] ^= QC32dn(k * 0x10, 11);} break;
			case 12:
	#pragma unroll 16
				for(int k=0;k<16;k++) { a[(k*2)+0] ^= QC32up(k * 0x10, 12); a[(k*2)+1] ^= QC32dn(k * 0x10, 12);} break;
			case 13:
	#pragma unroll 16
				for(int k=0;k<16;k++) { a[(k*2)+0] ^= QC32up(k * 0x10, 13); a[(k*2)+1] ^= QC32dn(k * 0x10, 13);} break;
		}

        // RBTT
#pragma unroll 16
        for(int k=0;k<32;k+=2)
        {
            uint32_t t2_0  = B32_0(a[(k +  2) & 0x1f]), t1_0  = B32_0(a[(k +  1) & 0x1f]);
            uint32_t t6_1  = B32_1(a[(k +  6) & 0x1f]), t5_1  = B32_1(a[(k +  5) & 0x1f]);
            uint32_t t10_2 = B32_2(a[(k + 10) & 0x1f]), t9_2  = B32_2(a[(k +  9) & 0x1f]);
            uint32_t t22_3 = B32_3(a[(k + 22) & 0x1f]), t13_3 = B32_3(a[(k + 13) & 0x1f]);
        
            t[k + 0] =  T0up( t2_0 ) ^ T1up( t6_1 ) ^ T2up( t10_2 ) ^ T3up( t22_3 ) ^ 
                        T0dn( t1_0 ) ^ T1dn( t5_1 ) ^ T2dn(  t9_2 ) ^ T3dn( t13_3 );

            t[k + 1] =  T0dn( t2_0 ) ^ T1dn( t6_1 ) ^ T2dn( t10_2 ) ^ T3dn( t22_3 ) ^ 
                        T0up( t1_0 ) ^ T1up( t5_1 ) ^ T2up(  t9_2 ) ^ T3up( t13_3 );
        }
#pragma unroll 32
        for(int k=0;k<32;k++)
            a[k] = t[k];
    }
}

__global__ void 
myriadgroestl_gpu_hash(int threads, uint32_t startNounce, uint32_t *resNounce)
{
#if USE_SHARED
	extern __shared__ char mixtabs[];

	if (threadIdx.x < 256)
	{
		*((uint32_t*)mixtabs + (    threadIdx.x)) = tex1Dfetch(t0up1, threadIdx.x);
		*((uint32_t*)mixtabs + (256+threadIdx.x)) = tex1Dfetch(t0dn1, threadIdx.x);
		*((uint32_t*)mixtabs + (512+threadIdx.x)) = tex1Dfetch(t1up1, threadIdx.x);
		*((uint32_t*)mixtabs + (768+threadIdx.x)) = tex1Dfetch(t1dn1, threadIdx.x);
		*((uint32_t*)mixtabs + (1024+threadIdx.x)) = tex1Dfetch(t2up1, threadIdx.x);
		*((uint32_t*)mixtabs + (1280+threadIdx.x)) = tex1Dfetch(t2dn1, threadIdx.x);
		*((uint32_t*)mixtabs + (1536+threadIdx.x)) = tex1Dfetch(t3up1, threadIdx.x);
		*((uint32_t*)mixtabs + (1792+threadIdx.x)) = tex1Dfetch(t3dn1, threadIdx.x);
	}

	__syncthreads();
#endif

    int thread = (blockDim.x * blockIdx.x + threadIdx.x);
    if (thread < threads)
    {
		// GROESTL
		uint32_t message[32];
		uint32_t state[32];

#pragma unroll 32
		for(int k=0;k<32;k++) message[k] = myriadgroestl_gpu_msg[k];

		uint32_t nounce = startNounce + thread;
		message[19] = SWAB32(nounce);

#pragma unroll 32
		for(int u=0;u<32;u++) state[u] = message[u];
		state[31] ^= 0x20000;

		// Perm
#if USE_SHARED
		myriadgroestl_perm_P(state, mixtabs);
		state[31] ^= 0x20000;
		myriadgroestl_perm_Q(message, mixtabs);
#else
		myriadgroestl_perm_P(state, NULL);
		state[31] ^= 0x20000;
		myriadgroestl_perm_Q(message, NULL);
#endif
#pragma unroll 32
		for(int u=0;u<32;u++) state[u] ^= message[u];

#pragma unroll 32
		for(int u=0;u<32;u++) message[u] = state[u];

#if USE_SHARED
		myriadgroestl_perm_P(message, mixtabs);
#else
		myriadgroestl_perm_P(message, NULL);
#endif

#pragma unroll 32
		for(int u=0;u<32;u++) state[u] ^= message[u];

        uint32_t out_state[16];
#pragma unroll 16
		for(int u=0;u<16;u++) out_state[u] = state[u+16];
        myriadgroestl_gpu_sha256(out_state);
        
        int i, position = -1;
        bool rc = true;

#pragma unroll 8
        for (i = 7; i >= 0; i--) {
            if (out_state[i] > pTarget[i]) {
                if(position < i) {
                    position = i;
                    rc = false;
                }
             }
             if (out_state[i] < pTarget[i]) {
                if(position < i) {
                    position = i;
                    rc = true;
                }
             }
        }

        if(rc == true)
            if(resNounce[0] > nounce)
                resNounce[0] = nounce;
    }
}

#define texDef(texname, texmem, texsource, texsize) \
	unsigned int *texmem; \
	hipMalloc(&texmem, texsize); \
	hipMemcpy(texmem, texsource, texsize, hipMemcpyHostToDevice); \
	texname.normalized = 0; \
	texname.filterMode = hipFilterModePoint; \
	texname.addressMode[0] = hipAddressModeClamp; \
	{ hipChannelFormatDesc channelDesc = hipCreateChannelDesc<unsigned int>(); \
	  hipBindTexture(NULL, &texname, texmem, &channelDesc, texsize ); } \

// Setup-Funktionen
__host__ void myriadgroestl_cpu_init(int thr_id, int threads)
{
	hipSetDevice(device_map[thr_id]);
	
	hipMemcpyToSymbol(HIP_SYMBOL(	myr_sha256_gpu_hashTable),
						myr_sha256_cpu_hashTable,
						sizeof(uint32_t) * 8 );

	hipMemcpyToSymbol(HIP_SYMBOL(	myr_sha256_gpu_constantTable),
						myr_sha256_cpu_constantTable,
						sizeof(uint32_t) * 64 );

    hipGetDeviceProperties(&props, device_map[thr_id]);

	// Texturen mit obigem Makro initialisieren
	texDef(t0up1, d_T0up, T0up_cpu, sizeof(uint32_t)*256);
	texDef(t0dn1, d_T0dn, T0dn_cpu, sizeof(uint32_t)*256);
	texDef(t1up1, d_T1up, T1up_cpu, sizeof(uint32_t)*256);
	texDef(t1dn1, d_T1dn, T1dn_cpu, sizeof(uint32_t)*256);
	texDef(t2up1, d_T2up, T2up_cpu, sizeof(uint32_t)*256);
	texDef(t2dn1, d_T2dn, T2dn_cpu, sizeof(uint32_t)*256);
	texDef(t3up1, d_T3up, T3up_cpu, sizeof(uint32_t)*256);
	texDef(t3dn1, d_T3dn, T3dn_cpu, sizeof(uint32_t)*256);

    // Speicher f�r Gewinner-Nonce belegen
    hipMalloc(&d_resultNonce[thr_id], sizeof(uint32_t)); 
}

__host__ void myriadgroestl_cpu_setBlock(int thr_id, void *data, void *pTargetIn)
{
    // Nachricht expandieren und setzen
    uint32_t msgBlock[32];

    memset(msgBlock, 0, sizeof(uint32_t) * 32);
    memcpy(&msgBlock[0], data, 80);

    // Erweitere die Nachricht auf den Nachrichtenblock (padding)
    // Unsere Nachricht hat 80 Byte
    msgBlock[20] = 0x80;
    msgBlock[31] = 0x01000000;

    // groestl512 braucht hierf�r keinen CPU-Code (die einzige Runde wird
    // auf der GPU ausgef�hrt)

    // Blockheader setzen (korrekte Nonce und Hefty Hash fehlen da drin noch)
    hipMemcpyToSymbol(HIP_SYMBOL( myriadgroestl_gpu_msg),
                        msgBlock,
                        128);

    hipMemset(d_resultNonce[thr_id], 0xFF, sizeof(uint32_t));
    hipMemcpyToSymbol(HIP_SYMBOL( pTarget),
                        pTargetIn,
                        sizeof(uint32_t) * 8 );
}

__host__ void myriadgroestl_cpu_hash(int thr_id, int threads, uint32_t startNounce, void *outputHashes, uint32_t *nounce)
{
	// Compute 3.x und 5.x Ger�te am besten mit 768 Threads ansteuern,
	// alle anderen mit 512 Threads.
	int threadsperblock = (props.major >= 3) ? 768 : 512;

    // berechne wie viele Thread Blocks wir brauchen
    dim3 grid((threads + threadsperblock-1)/threadsperblock);
    dim3 block(threadsperblock);

	// Gr��e des dynamischen Shared Memory Bereichs
#if USE_SHARED
	size_t shared_size = 8 * 256 * sizeof(uint32_t);
#else
	size_t shared_size = 0;
#endif

//    fprintf(stderr, "threads=%d, %d blocks, %d threads per block, %d bytes shared\n", threads, grid.x, block.x, shared_size);
    //fprintf(stderr, "ThrID: %d\n", thr_id);
    hipMemset(d_resultNonce[thr_id], 0xFF, sizeof(uint32_t));
    myriadgroestl_gpu_hash<<<grid, block, shared_size>>>(threads, startNounce, d_resultNonce[thr_id]);

    // Strategisches Sleep Kommando zur Senkung der CPU Last
    MyStreamSynchronize(NULL, 0, thr_id);

    hipMemcpy(nounce, d_resultNonce[thr_id], sizeof(uint32_t), hipMemcpyDeviceToHost);
}
