#include "hip/hip_runtime.h"

extern "C"
{
#include "sph/sph_keccak.h"
#include "sph/sph_blake.h"
#include "sph/sph_groestl.h"
#include "sph/sph_jh.h"
#include "sph/sph_skein.h"
}

#include "miner.h"
#include <stdint.h>

// aus cpu-miner.c
extern int device_map[8];
extern bool opt_benchmark;

// Speicher f�r Input/Output der verketteten Hashfunktionen
static uint32_t *d_hash[8];

extern void jackpot_keccak512_cpu_init(int thr_id, int threads);
extern void jackpot_keccak512_cpu_setBlock_88(void *pdata);
extern void jackpot_keccak512_cpu_hash_88(int thr_id, int threads, uint32_t startNounce, uint32_t *d_hash, int order);

extern void quark_check_cpu_init(int thr_id, int threads);
extern void quark_check_cpu_setTarget(const void *ptarget);
extern uint32_t quark_check_cpu_hash_64(int thr_id, int threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_inputHash, int order);

// Original jackpothash Funktion aus einem miner Quelltext
inline unsigned int jackpothash(void *state, const void *input)
{
    sph_blake512_context     ctx_blake;
    sph_groestl512_context   ctx_groestl;
    sph_jh512_context        ctx_jh;
    sph_keccak512_context    ctx_keccak;
    sph_skein512_context     ctx_skein;

    uint32_t hash[16];

    sph_keccak512_init(&ctx_keccak);
    sph_keccak512 (&ctx_keccak, input, 88);
    sph_keccak512_close(&ctx_keccak, hash);

    unsigned int round_mask = (
       (unsigned int)(((unsigned char *)input)[84]) <<  0 |
       (unsigned int)(((unsigned char *)input)[85]) <<  8 |
       (unsigned int)(((unsigned char *)input)[86]) << 16 |
       (unsigned int)(((unsigned char *)input)[87]) << 24 );
    unsigned int round_max  = hash[0] & round_mask;
    unsigned int round;
    for (round = 0; round < round_max; round++) {
        switch (hash[0] & 3) {
          case 0:
               sph_blake512_init(&ctx_blake);
               sph_blake512 (&ctx_blake, hash, 64);
               sph_blake512_close(&ctx_blake, hash);
               break;
          case 1:
               sph_groestl512_init(&ctx_groestl);
               sph_groestl512 (&ctx_groestl, hash, 64);
               sph_groestl512_close(&ctx_groestl, hash);
               break;
          case 2:
               sph_jh512_init(&ctx_jh);
               sph_jh512 (&ctx_jh, hash, 64);
               sph_jh512_close(&ctx_jh, hash);
               break;
          case 3:
               sph_skein512_init(&ctx_skein);
               sph_skein512 (&ctx_skein, hash, 64);
               sph_skein512_close(&ctx_skein, hash);
               break;
        }
    }
    memcpy(state, hash, 32);

    return round_max;
}


static int bit_population(uint32_t n){
  int c =0;
  while(n){
    c += n&1;
    n = n>>1;
  }
  return c;
}

extern "C" int scanhash_jackpot(int thr_id, uint32_t *pdata,
    const uint32_t *ptarget, uint32_t max_nonce,
    unsigned long *hashes_done)
{
	const uint32_t first_nonce = pdata[19];

	// TODO: entfernen f�r eine Release! Ist nur zum Testen!
	if (opt_benchmark) {
		((uint32_t*)ptarget)[7] = 0x00000f;
		((uint32_t*)pdata)[21] = 0x07000000;  // round_mask von 7 vorgeben
    }

	const uint32_t Htarg = ptarget[7];

	const int throughput = 256*4096; // 100;

	static bool init[8] = {0,0,0,0,0,0,0,0};
	if (!init[thr_id])
	{
		hipSetDevice(device_map[thr_id]);

		// Konstanten kopieren, Speicher belegen
		hipMalloc(&d_hash[thr_id], 16 * sizeof(uint32_t) * throughput);
		jackpot_keccak512_cpu_init(thr_id, throughput);
		quark_check_cpu_init(thr_id, throughput);
		init[thr_id] = true;
	}

	uint32_t endiandata[22];
	for (int k=0; k < 22; k++)
		be32enc(&endiandata[k], ((uint32_t*)pdata)[k]);

	unsigned int round_mask = (
		(unsigned int)(((unsigned char *)endiandata)[84]) <<  0 |
		(unsigned int)(((unsigned char *)endiandata)[85]) <<  8 |
		(unsigned int)(((unsigned char *)endiandata)[86]) << 16 |
		(unsigned int)(((unsigned char *)endiandata)[87]) << 24 );

	// Z�hlen wie viele Bits in round_mask gesetzt sind
	int bitcount = bit_population(round_mask);

	jackpot_keccak512_cpu_setBlock_88((void*)endiandata);
	quark_check_cpu_setTarget(ptarget);

	do {
		int order = 0;

		// erstes Blake512 Hash mit CUDA
		jackpot_keccak512_cpu_hash_88(thr_id, throughput, pdata[19], d_hash[thr_id], order++);

		// TODO: hier fehlen jetzt nat�rlich noch die anderen Hashrunden.
		// bei round_mask=7 haben wir eine 1:8 Chance, dass das Hash dennoch
		// die Kriterien erf�llt wenn hash[0] & round_mask  zuf�llig 0 ist.

		// Scan nach Gewinner Hashes auf der GPU
		uint32_t foundNonce = quark_check_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
		if  (foundNonce != 0xffffffff)
		{
			uint32_t vhash64[8];
			be32enc(&endiandata[19], foundNonce);

			// diese jackpothash Funktion gibt die Zahl der zus�tzlichen Runden zur�ck
			unsigned int rounds = jackpothash(vhash64, endiandata);

			// wir akzeptieren nur solche Hashes wo ausschliesslich Keccak verwendet wurde
			if (rounds == 0) {
				if ((vhash64[7]<=Htarg) && fulltest(vhash64, ptarget)) {

					pdata[19] = foundNonce;
					*hashes_done = (foundNonce - first_nonce + 1) / (1 << bitcount);
					return 1;
				} else {
					applog(LOG_INFO, "GPU #%d: result for nonce $%08X does not validate on CPU (%d rounds)!", thr_id, foundNonce, rounds);
				}
			}
		}

		pdata[19] += throughput;

	} while (pdata[19] < max_nonce && !work_restart[thr_id].restart);

	*hashes_done = (pdata[19] - first_nonce + 1) / (1 << bitcount);
	return 0;
}
