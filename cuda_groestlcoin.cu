#include "hip/hip_runtime.h"
// Auf Groestlcoin spezialisierte Version von Groestl

#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <memory.h>

#define USE_SHARED 1

extern hipError_t MyStreamSynchronize(hipStream_t stream, int situation, int thr_id);

// Folgende Definitionen sp�ter durch header ersetzen
typedef unsigned char uint8_t;
typedef unsigned int uint32_t;
typedef unsigned long long uint64_t;

// globaler Speicher f�r alle HeftyHashes aller Threads
__constant__ uint32_t pTarget[8]; // Single GPU
extern uint32_t *d_resultNonce[8];

// globaler Speicher f�r unsere Ergebnisse
uint32_t *d_hashGROESTLCOINoutput[8];

__constant__ uint32_t groestlcoin_gpu_state[32];
__constant__ uint32_t groestlcoin_gpu_msg[32];
__constant__ uint32_t sha256coin_gpu_constantTable[64];
__constant__ uint32_t sha256coin_gpu_register[8];

#define SPH_T32(x)    ((x) & SPH_C32(0xFFFFFFFF))

#define PC32up(j, r)   ((uint32_t)((j) + (r)))
#define PC32dn(j, r)   0
#define QC32up(j, r)   0xFFFFFFFF
#define QC32dn(j, r)   (((uint32_t)(r) << 24) ^ SPH_T32(~((uint32_t)(j) << 24)))

#define B32_0(x)    ((x) & 0xFF)
#define B32_1(x)    (((x) >> 8) & 0xFF)
#define B32_2(x)    (((x) >> 16) & 0xFF)
#define B32_3(x)    ((x) >> 24)

#define SPH_C32(x)	((uint32_t)(x ## U))
#define C32e(x)     ((SPH_C32(x) >> 24) \
                    | ((SPH_C32(x) >>  8) & SPH_C32(0x0000FF00)) \
                    | ((SPH_C32(x) <<  8) & SPH_C32(0x00FF0000)) \
                    | ((SPH_C32(x) << 24) & SPH_C32(0xFF000000)))

#if USE_SHARED
#define T0up(x) (*((uint32_t*)mixtabs + (    (x))))
#define T0dn(x) (*((uint32_t*)mixtabs + (256+(x))))
#define T1up(x) (*((uint32_t*)mixtabs + (512+(x))))
#define T1dn(x) (*((uint32_t*)mixtabs + (768+(x))))
#define T2up(x) (*((uint32_t*)mixtabs + (1024+(x))))
#define T2dn(x) (*((uint32_t*)mixtabs + (1280+(x))))
#define T3up(x) (*((uint32_t*)mixtabs + (1536+(x))))
#define T3dn(x) (*((uint32_t*)mixtabs + (1792+(x))))
#else
#define T0up(x) tex1Dfetch(t0up1, x)
#define T0dn(x) tex1Dfetch(t0dn1, x)
#define T1up(x) tex1Dfetch(t1up1, x)
#define T1dn(x) tex1Dfetch(t1dn1, x)
#define T2up(x) tex1Dfetch(t2up1, x)
#define T2dn(x) tex1Dfetch(t2dn1, x)
#define T3up(x) tex1Dfetch(t3up1, x)
#define T3dn(x) tex1Dfetch(t3dn1, x)
#endif
texture<unsigned int, 1, hipReadModeElementType> t0up1;
texture<unsigned int, 1, hipReadModeElementType> t0dn1;
texture<unsigned int, 1, hipReadModeElementType> t1up1;
texture<unsigned int, 1, hipReadModeElementType> t1dn1;
texture<unsigned int, 1, hipReadModeElementType> t2up1;
texture<unsigned int, 1, hipReadModeElementType> t2dn1;
texture<unsigned int, 1, hipReadModeElementType> t3up1;
texture<unsigned int, 1, hipReadModeElementType> t3dn1;

extern uint32_t T0up_cpu[];
extern uint32_t T0dn_cpu[];
extern uint32_t T1up_cpu[];
extern uint32_t T1dn_cpu[];
extern uint32_t T2up_cpu[];
extern uint32_t T2dn_cpu[];
extern uint32_t T3up_cpu[];
extern uint32_t T3dn_cpu[];

#define S(x, n)			(((x) >> (n)) | ((x) << (32 - (n))))
#define R(x, n)			((x) >> (n))
#define Ch(x, y, z)		((x & (y ^ z)) ^ z)
#define Maj(x, y, z)	((x & (y | z)) | (y & z))
#define S0(x)			(S(x, 2) ^ S(x, 13) ^ S(x, 22))
#define S1(x)			(S(x, 6) ^ S(x, 11) ^ S(x, 25))
#define s0(x)			(S(x, 7) ^ S(x, 18) ^ R(x, 3))
#define s1(x)			(S(x, 17) ^ S(x, 19) ^ R(x, 10))

#define SWAB32(x)		( ((x & 0x000000FF) << 24) | ((x & 0x0000FF00) << 8) | ((x & 0x00FF0000) >> 8) | ((x & 0xFF000000) >> 24) )


__device__ void groestlcoin_perm_P(uint32_t *a, char *mixtabs)
{
	uint32_t t[32];

//#pragma unroll 14
	for(int r=0;r<14;r++)
	{
#pragma unroll 16
		for(int k=0;k<16;k++)
		{
			a[(k*2)+0] ^= PC32up(k * 0x10, r);
			//a[(k<<1)+1] ^= PC32dn(k * 0x10, r);
		}

		// RBTT
#pragma unroll 16
		for(int k=0;k<32;k+=2)
		{
			t[k + 0] =	T0up( B32_0(a[k & 0x1f]) ) ^ 
						T1up( B32_1(a[(k + 2) & 0x1f]) ) ^ 
						T2up( B32_2(a[(k + 4) & 0x1f]) ) ^ 
						T3up( B32_3(a[(k + 6) & 0x1f]) ) ^ 
						T0dn( B32_0(a[(k + 9) & 0x1f]) ) ^ 
						T1dn( B32_1(a[(k + 11) & 0x1f]) ) ^ 
						T2dn( B32_2(a[(k + 13) & 0x1f]) ) ^ 
						T3dn( B32_3(a[(k + 23) & 0x1f]) );

			t[k + 1] =	T0dn( B32_0(a[k & 0x1f]) ) ^ 
						T1dn( B32_1(a[(k + 2) & 0x1f]) ) ^ 
						T2dn( B32_2(a[(k + 4) & 0x1f]) ) ^ 
						T3dn( B32_3(a[(k + 6) & 0x1f]) ) ^ 
						T0up( B32_0(a[(k + 9) & 0x1f]) ) ^ 
						T1up( B32_1(a[(k + 11) & 0x1f]) ) ^ 
						T2up( B32_2(a[(k + 13) & 0x1f]) ) ^ 
						T3up( B32_3(a[(k + 23) & 0x1f]) );
		}
#pragma unroll 32
		for(int k=0;k<32;k++)
			a[k] = t[k];
	}
}

__device__ void groestlcoin_perm_Q(uint32_t *a, char *mixtabs)
{	
//#pragma unroll 14
	for(int r=0;r<14;r++)
	{
		uint32_t t[32];

#pragma unroll 16
		for(int k=0;k<16;k++)
		{
			a[(k*2)+0] ^= QC32up(k * 0x10, r);
			a[(k*2)+1] ^= QC32dn(k * 0x10, r);
		}

		// RBTT
#pragma unroll 16
		for(int k=0;k<32;k+=2)
		{
			t[k + 0] =	T0up( B32_0(a[(k + 2) & 0x1f]) ) ^ 
						T1up( B32_1(a[(k + 6) & 0x1f]) ) ^ 
						T2up( B32_2(a[(k + 10) & 0x1f]) ) ^ 
						T3up( B32_3(a[(k + 22) & 0x1f]) ) ^ 
						T0dn( B32_0(a[(k + 1) & 0x1f]) ) ^ 
						T1dn( B32_1(a[(k + 5) & 0x1f]) ) ^ 
						T2dn( B32_2(a[(k + 9) & 0x1f]) ) ^ 
						T3dn( B32_3(a[(k + 13) & 0x1f]) );

			t[k + 1] =	T0dn( B32_0(a[(k + 2) & 0x1f]) ) ^ 
						T1dn( B32_1(a[(k + 6) & 0x1f]) ) ^ 
						T2dn( B32_2(a[(k + 10) & 0x1f]) ) ^ 
						T3dn( B32_3(a[(k + 22) & 0x1f]) ) ^ 
						T0up( B32_0(a[(k + 1) & 0x1f]) ) ^ 
						T1up( B32_1(a[(k + 5) & 0x1f]) ) ^ 
						T2up( B32_2(a[(k + 9) & 0x1f]) ) ^ 
						T3up( B32_3(a[(k + 13) & 0x1f]) );
		}
#pragma unroll 32
		for(int k=0;k<32;k++)
			a[k] = t[k];
	}
}
#if USE_SHARED
__global__ void  __launch_bounds__(256) 
#else
__global__ void 
#endif

 groestlcoin_gpu_hash(int threads, uint32_t startNounce, void *outputHash, uint32_t *resNounce)
{
#if USE_SHARED
	extern __shared__ char mixtabs[];

	*((uint32_t*)mixtabs + (    threadIdx.x)) = tex1Dfetch(t0up1, threadIdx.x);
	*((uint32_t*)mixtabs + (256+threadIdx.x)) = tex1Dfetch(t0dn1, threadIdx.x);
	*((uint32_t*)mixtabs + (512+threadIdx.x)) = tex1Dfetch(t1up1, threadIdx.x);
	*((uint32_t*)mixtabs + (768+threadIdx.x)) = tex1Dfetch(t1dn1, threadIdx.x);
	*((uint32_t*)mixtabs + (1024+threadIdx.x)) = tex1Dfetch(t2up1, threadIdx.x);
	*((uint32_t*)mixtabs + (1280+threadIdx.x)) = tex1Dfetch(t2dn1, threadIdx.x);
	*((uint32_t*)mixtabs + (1536+threadIdx.x)) = tex1Dfetch(t3up1, threadIdx.x);
	*((uint32_t*)mixtabs + (1792+threadIdx.x)) = tex1Dfetch(t3dn1, threadIdx.x);

	__syncthreads();
#endif

	int thread = (blockDim.x * blockIdx.x + threadIdx.x);
	if (thread < threads)
	{
	/////
	///// Lieber groestl, mach, dass es abgeht!!!
	/////
		// GROESTL
		uint32_t message[32];
		uint32_t state[32];
		uint32_t g[32];


#pragma unroll 32
		for(int k=0;k<32;k++)
		{
                        // TODO: die Vorbelegung mit Nullen braucht nicht zwingend aus dem
                        //       constant Memory zu lesen. Das ist Verschwendung von Bandbreite.
			state[k] = groestlcoin_gpu_state[k];
			message[k] = groestlcoin_gpu_msg[k];
		}

		uint32_t nounce = startNounce + thread;
		message[19] = SWAB32(nounce);

#pragma unroll 32
		for(int u=0;u<32;u++)
			g[u] = message[u] ^ state[u];  // TODO: state ist fast ueberall 0.

		// Perm
#if USE_SHARED
		groestlcoin_perm_P(g, mixtabs);        // TODO: g[] entspricht fast genau message[]
		groestlcoin_perm_Q(message, mixtabs);  //       kann man das ausnutzen?
#else
		groestlcoin_perm_P(g, NULL);
		groestlcoin_perm_Q(message, NULL);
#endif
		
#pragma unroll 32
		for(int u=0;u<32;u++)
		{
                        // TODO: kann man evtl. das xor mit g[u] vorziehen hinter die groestlcoin_perm_P Funktion
                        //       was den Registerbedarf senken koennte?
			state[u] ^= g[u] ^ message[u];
			g[u] = state[u];
		}

#if USE_SHARED
		groestlcoin_perm_P(g, mixtabs);
#else
		groestlcoin_perm_P(g, NULL);
#endif

#pragma unroll 32
		for(int u=0;u<32;u++)
			state[u] ^= g[u];

		////
		//// 2. Runde groestl
		////
#pragma unroll 16
		for(int k=0;k<16;k++)
			message[k] = state[k + 16];

#pragma unroll 32
		for(int k=0;k<32;k++)
			state[k] = groestlcoin_gpu_state[k];

#pragma unroll 16
		for(int k=0;k<16;k++)
			message[k+16] = 0;

		message[16] = 0x80;		
		message[31] = 0x01000000;

#pragma unroll 32
		for(int u=0;u<32;u++)
			g[u] = message[u] ^ state[u];

		// Perm
#if USE_SHARED
		groestlcoin_perm_P(g, mixtabs);
		groestlcoin_perm_Q(message, mixtabs);
#else
		groestlcoin_perm_P(g, NULL);
		groestlcoin_perm_Q(message, NULL);
#endif
		
#pragma unroll 32
		for(int u=0;u<32;u++)
		{
			state[u] ^= g[u] ^ message[u];
			g[u] = state[u];
		}

#if USE_SHARED
		groestlcoin_perm_P(g, mixtabs);
#else
		groestlcoin_perm_P(g, NULL);
#endif

#pragma unroll 32
		for(int u=0;u<32;u++)
			state[u] ^= g[u];
		
/*
	#pragma unroll 8
		for(int k=0;k<8;k++)
			hash[k] = state[k+16];
*/

		// kopiere Ergebnis
		/*
#pragma unroll 16
		for(int k=0;k<16;k++)
			((uint32_t*)outputHash)[16*thread+k] = state[k + 16];
			*/
		int i;
		bool rc = true;
	
		for (i = 7; i >= 0; i--) {
			if (state[i+16] > pTarget[i]) {
				rc = false;
				break;
			}
			if (state[i+16] < pTarget[i]) {
				rc = true;
				break;
			}
		}

		if(rc == true)
		{
			if(resNounce[0] > nounce)
			{
				resNounce[0] = nounce;
				/*
				#pragma unroll 8
				for(int k=0;k<8;k++)					
					((uint32_t*)outputHash)[k] = (hash[k]);
				*/
			}
		}

	}
}

#define texDef(texname, texmem, texsource, texsize) \
	unsigned int *texmem; \
	hipMalloc(&texmem, texsize); \
	hipMemcpy(texmem, texsource, texsize, hipMemcpyHostToDevice); \
	texname.normalized = 0; \
	texname.filterMode = hipFilterModePoint; \
	texname.addressMode[0] = hipAddressModeClamp; \
	{ hipChannelFormatDesc channelDesc = hipCreateChannelDesc<unsigned int>(); \
	  hipBindTexture(NULL, &texname, texmem, &channelDesc, texsize ); } \

// Setup-Funktionen
__host__ void groestlcoin_cpu_init(int thr_id, int threads)
{
	hipSetDevice(thr_id);
	hipDeviceSetCacheConfig( hipFuncCachePreferShared );
// Texturen mit obigem Makro initialisieren
	texDef(t0up1, d_T0up, T0up_cpu, sizeof(uint32_t)*256);
	texDef(t0dn1, d_T0dn, T0dn_cpu, sizeof(uint32_t)*256);
	texDef(t1up1, d_T1up, T1up_cpu, sizeof(uint32_t)*256);
	texDef(t1dn1, d_T1dn, T1dn_cpu, sizeof(uint32_t)*256);
	texDef(t2up1, d_T2up, T2up_cpu, sizeof(uint32_t)*256);
	texDef(t2dn1, d_T2dn, T2dn_cpu, sizeof(uint32_t)*256);
	texDef(t3up1, d_T3up, T3up_cpu, sizeof(uint32_t)*256);
	texDef(t3dn1, d_T3dn, T3dn_cpu, sizeof(uint32_t)*256);

	// setze register 
        // TODO: fast vollstaendige Vorbelegung mit Nullen.
        //       da besteht doch Optimierungspotenzial im GPU Kernel
        //       denn mit Nullen braucht man nicht wirklich rechnen.
	uint32_t groestl_state_init[32];
	memset(groestl_state_init, 0, sizeof(uint32_t) * 32);
	groestl_state_init[31] = 0x20000;

	// state speichern
	hipMemcpyToSymbol(HIP_SYMBOL(	groestlcoin_gpu_state),
						groestl_state_init,
						128);

	hipMalloc(&d_resultNonce[thr_id], sizeof(uint32_t)); 

	// Speicher f�r alle Ergebnisse belegen (nur f�r Debug)
	hipMalloc(&d_hashGROESTLCOINoutput[thr_id], 8 * sizeof(uint32_t) * threads);
}

__host__ void groestlcoin_cpu_setBlock(int thr_id, void *data, void *pTargetIn)
{
	// Nachricht expandieren und setzen
	uint32_t msgBlock[32];

	memset(msgBlock, 0, sizeof(uint32_t) * 32);
	memcpy(&msgBlock[0], data, 80);

	// Erweitere die Nachricht auf den Nachrichtenblock (padding)
	// Unsere Nachricht hat 80 Byte
	msgBlock[20] = 0x80;
	msgBlock[31] = 0x01000000;

	// groestl512 braucht hierf�r keinen CPU-Code (die einzige Runde wird
	// auf der GPU ausgef�hrt)

	// Blockheader setzen (korrekte Nonce und Hefty Hash fehlen da drin noch)
	hipMemcpyToSymbol(HIP_SYMBOL(	groestlcoin_gpu_msg),
						msgBlock,
						128);

	hipMemset(d_resultNonce[thr_id], 0xFF, sizeof(uint32_t));
	hipMemcpyToSymbol(HIP_SYMBOL(	pTarget),
						pTargetIn,
						sizeof(uint32_t) * 8 );
}

__host__ void groestlcoin_cpu_hash(int thr_id, int threads, uint32_t startNounce, void *outputHashes, uint32_t *nounce)
{
#if USE_SHARED
	const int threadsperblock = 256; // Alignment mit mixtab Gr�sse. NICHT �NDERN
#else
	const int threadsperblock = 512; // so einstellen wie gew�nscht ;-)
#endif

	// berechne wie viele Thread Blocks wir brauchen
	dim3 grid((threads + threadsperblock-1)/threadsperblock);
	dim3 block(threadsperblock);

	// Gr��e des dynamischen Shared Memory Bereichs (abh�ngig von der Threadanzahl)
#if USE_SHARED
	size_t shared_size = 8 * 256 * sizeof(uint32_t);
#else
	size_t shared_size = 0;
#endif

//	fprintf(stderr, "threads=%d, %d blocks, %d threads per block, %d bytes shared\n", threads, grid.x, block.x, shared_size);
	//fprintf(stderr, "ThrID: %d\n", thr_id);
	hipMemset(d_resultNonce[thr_id], 0xFF, sizeof(uint32_t));
	groestlcoin_gpu_hash<<<grid, block, shared_size>>>(threads, startNounce, d_hashGROESTLCOINoutput[thr_id], d_resultNonce[thr_id]);

	// Strategisches Sleep Kommando zur Senkung der CPU Last
	MyStreamSynchronize(NULL, 0, thr_id);

	hipMemcpy(nounce, d_resultNonce[thr_id], sizeof(uint32_t), hipMemcpyDeviceToHost);

	/// Debug
	//hipMemcpy(outputHashes, d_hashGROESTLCOINoutput[thr_id], 8 * sizeof(uint32_t) * threads, hipMemcpyDeviceToHost);

	// Nounce
	//hipMemcpy(nounce, d_resultNonce[thr_id], sizeof(uint32_t), hipMemcpyDeviceToHost);
}
