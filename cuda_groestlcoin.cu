#include "hip/hip_runtime.h"
// Auf Groestlcoin spezialisierte Version von Groestl

#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <memory.h>

// it's unfortunate that this is a compile time constant.
#define MAXWELL_OR_FERMI 1

// aus cpu-miner.c
extern int device_map[8];

// aus heavy.cu
extern hipError_t MyStreamSynchronize(hipStream_t stream, int situation, int thr_id);

// Folgende Definitionen sp�ter durch header ersetzen
typedef unsigned char uint8_t;
typedef unsigned int uint32_t;
typedef unsigned long long uint64_t;

// diese Struktur wird in der Init Funktion angefordert
static hipDeviceProp_t props;

__constant__ uint32_t pTarget[8]; // Single GPU
extern uint32_t *d_resultNonce[8];

__constant__ uint32_t groestlcoin_gpu_msg[32];

#define SPH_C32(x)    ((uint32_t)(x ## U))
#define SPH_T32(x)    ((x) & SPH_C32(0xFFFFFFFF))

#define PC32up(j, r)   ((uint32_t)((j) + (r)))
#define PC32dn(j, r)   0
#define QC32up(j, r)   0xFFFFFFFF
#define QC32dn(j, r)   (((uint32_t)(r) << 24) ^ SPH_T32(~((uint32_t)(j) << 24)))

#define B32_0(x)    __byte_perm(x, 0, 0x4440)
//((x) & 0xFF)
#define B32_1(x)    __byte_perm(x, 0, 0x4441)
//(((x) >> 8) & 0xFF)
#define B32_2(x)    __byte_perm(x, 0, 0x4442)
//(((x) >> 16) & 0xFF)
#define B32_3(x)    __byte_perm(x, 0, 0x4443)
//((x) >> 24)

#if MAXWELL_OR_FERMI
#define USE_SHARED 1
// Maxwell and Fermi cards get the best speed with SHARED access it seems.
#if USE_SHARED
#define T0up(x) (*((uint32_t*)mixtabs + (    (x))))
#define T0dn(x) (*((uint32_t*)mixtabs + (256+(x))))
#define T1up(x) (*((uint32_t*)mixtabs + (512+(x))))
#define T1dn(x) (*((uint32_t*)mixtabs + (768+(x))))
#define T2up(x) (*((uint32_t*)mixtabs + (1024+(x))))
#define T2dn(x) (*((uint32_t*)mixtabs + (1280+(x))))
#define T3up(x) (*((uint32_t*)mixtabs + (1536+(x))))
#define T3dn(x) (*((uint32_t*)mixtabs + (1792+(x))))
#else
#define T0up(x) tex1Dfetch(t0up1, x)
#define T0dn(x) tex1Dfetch(t0dn1, x)
#define T1up(x) tex1Dfetch(t1up1, x)
#define T1dn(x) tex1Dfetch(t1dn1, x)
#define T2up(x) tex1Dfetch(t2up1, x)
#define T2dn(x) tex1Dfetch(t2dn1, x)
#define T3up(x) tex1Dfetch(t3up1, x)
#define T3dn(x) tex1Dfetch(t3dn1, x)
#endif
#else
#define USE_SHARED 1
// a healthy mix between shared and textured access provides the highest speed on Compute 3.0 and 3.5!
#define T0up(x) (*((uint32_t*)mixtabs + (    (x))))
#define T0dn(x) tex1Dfetch(t0dn1, x)
#define T1up(x) tex1Dfetch(t1up1, x)
#define T1dn(x) (*((uint32_t*)mixtabs + (768+(x))))
#define T2up(x) tex1Dfetch(t2up1, x)
#define T2dn(x) (*((uint32_t*)mixtabs + (1280+(x))))
#define T3up(x) (*((uint32_t*)mixtabs + (1536+(x))))
#define T3dn(x) tex1Dfetch(t3dn1, x)
#endif

texture<unsigned int, 1, hipReadModeElementType> t0up1;
texture<unsigned int, 1, hipReadModeElementType> t0dn1;
texture<unsigned int, 1, hipReadModeElementType> t1up1;
texture<unsigned int, 1, hipReadModeElementType> t1dn1;
texture<unsigned int, 1, hipReadModeElementType> t2up1;
texture<unsigned int, 1, hipReadModeElementType> t2dn1;
texture<unsigned int, 1, hipReadModeElementType> t3up1;
texture<unsigned int, 1, hipReadModeElementType> t3dn1;

extern uint32_t T0up_cpu[];
extern uint32_t T0dn_cpu[];
extern uint32_t T1up_cpu[];
extern uint32_t T1dn_cpu[];
extern uint32_t T2up_cpu[];
extern uint32_t T2dn_cpu[];
extern uint32_t T3up_cpu[];
extern uint32_t T3dn_cpu[];

#define SWAB32(x)		( ((x & 0x000000FF) << 24) | ((x & 0x0000FF00) << 8) | ((x & 0x00FF0000) >> 8) | ((x & 0xFF000000) >> 24) )


__device__ __forceinline__ void groestlcoin_perm_P(uint32_t *a, char *mixtabs)
{
	uint32_t t[32];

//#pragma unroll 14
	for(int r=0;r<14;r++)
	{
		switch(r)
		{
			case 0:
#pragma unroll 16
				for(int k=0;k<16;k++) a[(k*2)+0] ^= PC32up(k * 0x10, 0); break;
			case 1:
#pragma unroll 16
				for(int k=0;k<16;k++) a[(k*2)+0] ^= PC32up(k * 0x10, 1); break;
			case 2:
#pragma unroll 16
				for(int k=0;k<16;k++) a[(k*2)+0] ^= PC32up(k * 0x10, 2); break;
			case 3:
#pragma unroll 16
				for(int k=0;k<16;k++) a[(k*2)+0] ^= PC32up(k * 0x10, 3); break;
			case 4:
#pragma unroll 16
				for(int k=0;k<16;k++) a[(k*2)+0] ^= PC32up(k * 0x10, 4); break;
			case 5:
#pragma unroll 16
				for(int k=0;k<16;k++) a[(k*2)+0] ^= PC32up(k * 0x10, 5); break;
			case 6:
#pragma unroll 16
				for(int k=0;k<16;k++) a[(k*2)+0] ^= PC32up(k * 0x10, 6); break;
			case 7:
#pragma unroll 16
				for(int k=0;k<16;k++) a[(k*2)+0] ^= PC32up(k * 0x10, 7); break;
			case 8:
#pragma unroll 16
				for(int k=0;k<16;k++) a[(k*2)+0] ^= PC32up(k * 0x10, 8); break;
			case 9:
#pragma unroll 16
				for(int k=0;k<16;k++) a[(k*2)+0] ^= PC32up(k * 0x10, 9); break;
			case 10:
#pragma unroll 16
				for(int k=0;k<16;k++) a[(k*2)+0] ^= PC32up(k * 0x10, 10); break;
			case 11:
#pragma unroll 16
				for(int k=0;k<16;k++) a[(k*2)+0] ^= PC32up(k * 0x10, 11); break;
			case 12:
#pragma unroll 16
				for(int k=0;k<16;k++) a[(k*2)+0] ^= PC32up(k * 0x10, 12); break;
			case 13:
#pragma unroll 16
				for(int k=0;k<16;k++) a[(k*2)+0] ^= PC32up(k * 0x10, 13); break;
		}

        // RBTT
#pragma unroll 16
        for(int k=0;k<32;k+=2)
        {
            uint32_t t0_0 = B32_0(a[(k     ) & 0x1f]), t9_0  = B32_0(a[(k +  9) & 0x1f]);
            uint32_t t2_1 = B32_1(a[(k +  2) & 0x1f]), t11_1 = B32_1(a[(k + 11) & 0x1f]);
            uint32_t t4_2 = B32_2(a[(k +  4) & 0x1f]), t13_2 = B32_2(a[(k + 13) & 0x1f]);
            uint32_t t6_3 = B32_3(a[(k +  6) & 0x1f]), t23_3 = B32_3(a[(k + 23) & 0x1f]);
        
            t[k + 0] =  T0up( t0_0 ) ^ T1up(  t2_1 ) ^ T2up(  t4_2 ) ^ T3up(  t6_3 ) ^ 
                        T0dn( t9_0 ) ^ T1dn( t11_1 ) ^ T2dn( t13_2 ) ^ T3dn( t23_3 );

            t[k + 1] =  T0dn( t0_0 ) ^ T1dn(  t2_1 ) ^ T2dn(  t4_2 ) ^ T3dn(  t6_3 ) ^ 
                        T0up( t9_0 ) ^ T1up( t11_1 ) ^ T2up( t13_2 ) ^ T3up( t23_3 );
        }
#pragma unroll 32
        for(int k=0;k<32;k++)
            a[k] = t[k];
    }
}

__device__ __forceinline__ void groestlcoin_perm_Q(uint32_t *a, char *mixtabs)
{	
//#pragma unroll 14
	for(int r=0;r<14;r++)
	{
		uint32_t t[32];

		switch(r)
		{
			case 0:
	#pragma unroll 16
				for(int k=0;k<16;k++) { a[(k*2)+0] ^= QC32up(k * 0x10, 0); a[(k*2)+1] ^= QC32dn(k * 0x10, 0);} break;
			case 1:
	#pragma unroll 16
				for(int k=0;k<16;k++) { a[(k*2)+0] ^= QC32up(k * 0x10, 1); a[(k*2)+1] ^= QC32dn(k * 0x10, 1);} break;
			case 2:
	#pragma unroll 16
				for(int k=0;k<16;k++) { a[(k*2)+0] ^= QC32up(k * 0x10, 2); a[(k*2)+1] ^= QC32dn(k * 0x10, 2);} break;
			case 3:
	#pragma unroll 16
				for(int k=0;k<16;k++) { a[(k*2)+0] ^= QC32up(k * 0x10, 3); a[(k*2)+1] ^= QC32dn(k * 0x10, 3);} break;
			case 4:
	#pragma unroll 16
				for(int k=0;k<16;k++) { a[(k*2)+0] ^= QC32up(k * 0x10, 4); a[(k*2)+1] ^= QC32dn(k * 0x10, 4);} break;
			case 5:
	#pragma unroll 16
				for(int k=0;k<16;k++) { a[(k*2)+0] ^= QC32up(k * 0x10, 5); a[(k*2)+1] ^= QC32dn(k * 0x10, 5);} break;
			case 6:
	#pragma unroll 16
				for(int k=0;k<16;k++) { a[(k*2)+0] ^= QC32up(k * 0x10, 6); a[(k*2)+1] ^= QC32dn(k * 0x10, 6);} break;
			case 7:
	#pragma unroll 16
				for(int k=0;k<16;k++) { a[(k*2)+0] ^= QC32up(k * 0x10, 7); a[(k*2)+1] ^= QC32dn(k * 0x10, 7);} break;
			case 8:
	#pragma unroll 16
				for(int k=0;k<16;k++) { a[(k*2)+0] ^= QC32up(k * 0x10, 8); a[(k*2)+1] ^= QC32dn(k * 0x10, 8);} break;
			case 9:
	#pragma unroll 16
				for(int k=0;k<16;k++) { a[(k*2)+0] ^= QC32up(k * 0x10, 9); a[(k*2)+1] ^= QC32dn(k * 0x10, 9);} break;
			case 10:
	#pragma unroll 16
				for(int k=0;k<16;k++) { a[(k*2)+0] ^= QC32up(k * 0x10, 10); a[(k*2)+1] ^= QC32dn(k * 0x10, 10);} break;
			case 11:
	#pragma unroll 16
				for(int k=0;k<16;k++) { a[(k*2)+0] ^= QC32up(k * 0x10, 11); a[(k*2)+1] ^= QC32dn(k * 0x10, 11);} break;
			case 12:
	#pragma unroll 16
				for(int k=0;k<16;k++) { a[(k*2)+0] ^= QC32up(k * 0x10, 12); a[(k*2)+1] ^= QC32dn(k * 0x10, 12);} break;
			case 13:
	#pragma unroll 16
				for(int k=0;k<16;k++) { a[(k*2)+0] ^= QC32up(k * 0x10, 13); a[(k*2)+1] ^= QC32dn(k * 0x10, 13);} break;
		}

        // RBTT
#pragma unroll 16
        for(int k=0;k<32;k+=2)
        {
            uint32_t t2_0  = B32_0(a[(k +  2) & 0x1f]), t1_0  = B32_0(a[(k +  1) & 0x1f]);
            uint32_t t6_1  = B32_1(a[(k +  6) & 0x1f]), t5_1  = B32_1(a[(k +  5) & 0x1f]);
            uint32_t t10_2 = B32_2(a[(k + 10) & 0x1f]), t9_2  = B32_2(a[(k +  9) & 0x1f]);
            uint32_t t22_3 = B32_3(a[(k + 22) & 0x1f]), t13_3 = B32_3(a[(k + 13) & 0x1f]);
        
            t[k + 0] =  T0up( t2_0 ) ^ T1up( t6_1 ) ^ T2up( t10_2 ) ^ T3up( t22_3 ) ^ 
                        T0dn( t1_0 ) ^ T1dn( t5_1 ) ^ T2dn(  t9_2 ) ^ T3dn( t13_3 );

            t[k + 1] =  T0dn( t2_0 ) ^ T1dn( t6_1 ) ^ T2dn( t10_2 ) ^ T3dn( t22_3 ) ^ 
                        T0up( t1_0 ) ^ T1up( t5_1 ) ^ T2up(  t9_2 ) ^ T3up( t13_3 );
        }
#pragma unroll 32
        for(int k=0;k<32;k++)
            a[k] = t[k];
    }
}
#if USE_SHARED
__global__ void  /* __launch_bounds__(256) */
#else
__global__ void 
#endif

 groestlcoin_gpu_hash(int threads, uint32_t startNounce, uint32_t *resNounce)
{
#if USE_SHARED
	extern __shared__ char mixtabs[];

	if (threadIdx.x < 256)
	{
		*((uint32_t*)mixtabs + (    threadIdx.x)) = tex1Dfetch(t0up1, threadIdx.x);
		*((uint32_t*)mixtabs + (256+threadIdx.x)) = tex1Dfetch(t0dn1, threadIdx.x);
		*((uint32_t*)mixtabs + (512+threadIdx.x)) = tex1Dfetch(t1up1, threadIdx.x);
		*((uint32_t*)mixtabs + (768+threadIdx.x)) = tex1Dfetch(t1dn1, threadIdx.x);
		*((uint32_t*)mixtabs + (1024+threadIdx.x)) = tex1Dfetch(t2up1, threadIdx.x);
		*((uint32_t*)mixtabs + (1280+threadIdx.x)) = tex1Dfetch(t2dn1, threadIdx.x);
		*((uint32_t*)mixtabs + (1536+threadIdx.x)) = tex1Dfetch(t3up1, threadIdx.x);
		*((uint32_t*)mixtabs + (1792+threadIdx.x)) = tex1Dfetch(t3dn1, threadIdx.x);
	}

	__syncthreads();
#endif

	int thread = (blockDim.x * blockIdx.x + threadIdx.x);
	if (thread < threads)
	{
		// GROESTL
		uint32_t message[32];
		uint32_t state[32];

#pragma unroll 32
		for(int k=0;k<32;k++) message[k] = groestlcoin_gpu_msg[k];

		uint32_t nounce = startNounce + thread;
		message[19] = SWAB32(nounce);

#pragma unroll 32
		for(int u=0;u<32;u++) state[u] = message[u];
		state[31] ^= 0x20000;

		// Perm
#if USE_SHARED
		groestlcoin_perm_P(state, mixtabs);
		state[31] ^= 0x20000;
		groestlcoin_perm_Q(message, mixtabs);
#else
		groestlcoin_perm_P(state, NULL);
		state[31] ^= 0x20000;
		groestlcoin_perm_Q(message, NULL);
#endif
#pragma unroll 32
		for(int u=0;u<32;u++) state[u] ^= message[u];

#pragma unroll 32
		for(int u=0;u<32;u++) message[u] = state[u];

#if USE_SHARED
		groestlcoin_perm_P(message, mixtabs);
#else
		groestlcoin_perm_P(message, NULL);
#endif

#pragma unroll 32
		for(int u=0;u<32;u++) state[u] ^= message[u];

		////
		//// 2. Runde groestl
		////
#pragma unroll 16
		for(int k=0;k<16;k++) message[k] = state[k + 16];
#pragma unroll 14
		for(int k=1;k<15;k++)
			message[k+16] = 0;

		message[16] = 0x80;
		message[31] = 0x01000000;

#pragma unroll 32
		for(int u=0;u<32;u++)
			state[u] = message[u];
		state[31] ^= 0x20000;

		// Perm
#if USE_SHARED
		groestlcoin_perm_P(state, mixtabs);
		state[31] ^= 0x20000;
		groestlcoin_perm_Q(message, mixtabs);
#else
		groestlcoin_perm_P(state, NULL);
		state[31] ^= 0x20000;
		groestlcoin_perm_Q(message, NULL);
#endif
		
#pragma unroll 32
		for(int u=0;u<32;u++) state[u] ^= message[u];

#pragma unroll 32
		for(int u=0;u<32;u++) message[u] = state[u];

#if USE_SHARED
		groestlcoin_perm_P(message, mixtabs);
#else
		groestlcoin_perm_P(message, NULL);
#endif

#pragma unroll 32
		for(int u=0;u<32;u++) state[u] ^= message[u];

		// kopiere Ergebnis
		int i, position = -1;
		bool rc = true;

#pragma unroll 8
		for (i = 7; i >= 0; i--) {
			if (state[i+16] > pTarget[i]) {
				if(position < i) {
					position = i;
					rc = false;
				}
	 		}
	 		if (state[i+16] < pTarget[i]) {
				if(position < i) {
					position = i;
					rc = true;
				}
	 		}
		}

		if(rc == true)
			if(resNounce[0] > nounce)
				resNounce[0] = nounce;
	}
}

#define texDef(texname, texmem, texsource, texsize) \
	unsigned int *texmem; \
	hipMalloc(&texmem, texsize); \
	hipMemcpy(texmem, texsource, texsize, hipMemcpyHostToDevice); \
	texname.normalized = 0; \
	texname.filterMode = hipFilterModePoint; \
	texname.addressMode[0] = hipAddressModeClamp; \
	{ hipChannelFormatDesc channelDesc = hipCreateChannelDesc<unsigned int>(); \
	  hipBindTexture(NULL, &texname, texmem, &channelDesc, texsize ); } \

// Setup-Funktionen
__host__ void groestlcoin_cpu_init(int thr_id, int threads)
{
	hipSetDevice(device_map[thr_id]);

	hipGetDeviceProperties(&props, device_map[thr_id]);

	// Texturen mit obigem Makro initialisieren
	texDef(t0up1, d_T0up, T0up_cpu, sizeof(uint32_t)*256);
	texDef(t0dn1, d_T0dn, T0dn_cpu, sizeof(uint32_t)*256);
	texDef(t1up1, d_T1up, T1up_cpu, sizeof(uint32_t)*256);
	texDef(t1dn1, d_T1dn, T1dn_cpu, sizeof(uint32_t)*256);
	texDef(t2up1, d_T2up, T2up_cpu, sizeof(uint32_t)*256);
	texDef(t2dn1, d_T2dn, T2dn_cpu, sizeof(uint32_t)*256);
	texDef(t3up1, d_T3up, T3up_cpu, sizeof(uint32_t)*256);
	texDef(t3dn1, d_T3dn, T3dn_cpu, sizeof(uint32_t)*256);

	// Speicher f�r Gewinner-Nonce belegen
	hipMalloc(&d_resultNonce[thr_id], sizeof(uint32_t)); 
}

__host__ void groestlcoin_cpu_setBlock(int thr_id, void *data, void *pTargetIn)
{
	// Nachricht expandieren und setzen
	uint32_t msgBlock[32];

	memset(msgBlock, 0, sizeof(uint32_t) * 32);
	memcpy(&msgBlock[0], data, 80);

	// Erweitere die Nachricht auf den Nachrichtenblock (padding)
	// Unsere Nachricht hat 80 Byte
	msgBlock[20] = 0x80;
	msgBlock[31] = 0x01000000;

	// groestl512 braucht hierf�r keinen CPU-Code (die einzige Runde wird
	// auf der GPU ausgef�hrt)

	// Blockheader setzen (korrekte Nonce und Hefty Hash fehlen da drin noch)
	hipMemcpyToSymbol(HIP_SYMBOL(	groestlcoin_gpu_msg),
						msgBlock,
						128);

	hipMemset(d_resultNonce[thr_id], 0xFF, sizeof(uint32_t));
	hipMemcpyToSymbol(HIP_SYMBOL(	pTarget),
						pTargetIn,
						sizeof(uint32_t) * 8 );
}

__host__ void groestlcoin_cpu_hash(int thr_id, int threads, uint32_t startNounce, void *outputHashes, uint32_t *nounce)
{
	// Compute 3.x und 5.x Ger�te am besten mit 768 Threads ansteuern,
	// alle anderen mit 512 Threads.
	int threadsperblock = (props.major >= 3) ? 768 : 512;

	// berechne wie viele Thread Blocks wir brauchen
	dim3 grid((threads + threadsperblock-1)/threadsperblock);
	dim3 block(threadsperblock);

	// Gr��e des dynamischen Shared Memory Bereichs
#if USE_SHARED
	size_t shared_size = 8 * 256 * sizeof(uint32_t);
#else
	size_t shared_size = 0;
#endif

//	fprintf(stderr, "threads=%d, %d blocks, %d threads per block, %d bytes shared\n", threads, grid.x, block.x, shared_size);
	//fprintf(stderr, "ThrID: %d\n", thr_id);
	hipMemset(d_resultNonce[thr_id], 0xFF, sizeof(uint32_t));
	groestlcoin_gpu_hash<<<grid, block, shared_size>>>(threads, startNounce, d_resultNonce[thr_id]);

	// Strategisches Sleep Kommando zur Senkung der CPU Last
	MyStreamSynchronize(NULL, 0, thr_id);

	hipMemcpy(nounce, d_resultNonce[thr_id], sizeof(uint32_t), hipMemcpyDeviceToHost);
}
